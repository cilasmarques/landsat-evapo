#include "hip/hip_runtime.h"
#include "endmembers.h"

void compute_H0(float *net_radiation, float *soil_heat_flux, int height_band, int width_band, float *ho)
{
  for (int i = 0; i < height_band * width_band; i++)
    ho[i] = net_radiation[i] - soil_heat_flux[i];
};

void get_quartiles(float *target, float *v_quartile, int height_band, int width_band, float first_interval, float middle_interval, float last_interval)
{
  const int SIZE = height_band * width_band;
  float *target_values = (float *)malloc(sizeof(float) * SIZE);

  if (target_values == NULL)
    exit(15);

  int pos = 0;
  for (int i = 0; i < height_band * width_band; i++)
  {
    if (!isnan(target[i]) && !isinf(target[i]))
    {
      target_values[pos] = target[i];
      pos++;
    }
  }

  int first_index = static_cast<int>(floor(first_interval * pos));
  int middle_index = static_cast<int>(floor(middle_interval * pos));
  int last_index = static_cast<int>(floor(last_interval * pos));

  std::nth_element(target_values, target_values + first_index, target_values + pos);
  v_quartile[0] = target_values[first_index];

  std::nth_element(target_values, target_values + middle_index, target_values + pos);
  v_quartile[1] = target_values[middle_index];

  std::nth_element(target_values, target_values + last_index, target_values + pos);
  v_quartile[2] = target_values[last_index];

  free(target_values);
}

pair<Candidate, Candidate> getEndmembersSTEPP(float *ndvi, float *surface_temperature, float *albedo, float *net_radiation, float *soil_heat, int height_band, int width_band)
{
  vector<Candidate> hotCandidates;
  vector<Candidate> coldCandidates;

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles(ndvi, ndviQuartile.data(), height_band, width_band, 0.15, 0.85, 0.97);
  get_quartiles(albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature, tsQuartile.data(), height_band, width_band, 0.20, 0.85, 0.97);

  float *ho = (float *)malloc(sizeof(float) * height_band * width_band);
  compute_H0(net_radiation, soil_heat, height_band, width_band, ho);

  for (int i = 0; i < height_band * width_band; i++)
  {
    bool hotNDVI = !std::isnan(ndvi[i]) && ndvi[i] > 0.10 && ndvi[i] < ndviQuartile[0];
    bool hotAlbedo = !std::isnan(albedo[i]) && albedo[i] > albedoQuartile[1] && albedo[i] < albedoQuartile[2];
    bool hotTS = !std::isnan(surface_temperature[i]) && surface_temperature[i] > tsQuartile[1] && surface_temperature[i] < tsQuartile[2];

    bool coldNDVI = !std::isnan(ndvi[i]) && ndvi[i] > ndviQuartile[2];
    bool coldAlbedo = !std::isnan(surface_temperature[i]) && albedo[i] > albedoQuartile[0] && albedo[i] < albedoQuartile[1];
    bool coldTS = !std::isnan(albedo[i]) && surface_temperature[i] < tsQuartile[0];

    int line = i / width_band;
    int col = i % width_band;

    if (hotAlbedo && hotNDVI && hotTS) 
      hotCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);

    if (coldNDVI && coldAlbedo && coldTS)
      coldCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);
  }

  if (hotCandidates.empty() || coldCandidates.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(hotCandidates.begin(), hotCandidates.end(), compare_candidate_temperature);
  std::sort(coldCandidates.begin(), coldCandidates.end(), compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(hotCandidates.size() * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(coldCandidates.size() * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

pair<Candidate, Candidate> getEndmembersASEBAL(float *ndvi, float *surface_temperature, float *albedo, float *net_radiation, float *soil_heat, int height_band, int width_band)
{
  vector<Candidate> hotCandidates;
  vector<Candidate> coldCandidates;

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles(ndvi, ndviQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);
  get_quartiles(albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature, tsQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);

  float *ho = (float *)malloc(sizeof(float) * height_band * width_band);
  compute_H0(net_radiation, soil_heat, height_band, width_band, ho);

  for (int i = 0; i < height_band * width_band; i++)
  {
    bool hotAlbedo = !std::isnan(albedo[i]) && albedo[i] > albedoQuartile[1];
    bool hotNDVI = !std::isnan(ndvi[i]) && ndvi[i] > 0.10 && ndvi[i] < ndviQuartile[0];
    bool hotTS = !std::isnan(surface_temperature[i]) && surface_temperature[i] > tsQuartile[1];

    bool coldAlbedo = !std::isnan(albedo[i]) && albedo[i] < albedoQuartile[1];
    bool coldNDVI = !std::isnan(ndvi[i]) && ndvi[i] >= ndviQuartile[1];
    bool coldTS = !std::isnan(surface_temperature[i]) && surface_temperature[i] < tsQuartile[0];

    int line = i / width_band;
    int col = i % width_band;

    if (hotAlbedo && hotNDVI && hotTS) 
      hotCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);

    if (coldNDVI && coldAlbedo && coldTS)
      coldCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);
  }

  if (hotCandidates.empty() || coldCandidates.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(hotCandidates.begin(), hotCandidates.end(), compare_candidate_temperature);
  std::sort(coldCandidates.begin(), coldCandidates.end(), compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(hotCandidates.size() * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(coldCandidates.size() * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

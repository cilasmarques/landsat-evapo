#include "hip/hip_runtime.h"
#include "endmembers.h"
#include "filter.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

void get_quartiles_cuda(float *d_target, float *v_quartile, int height_band, int width_band,
                   float first_interval, float middle_interval, float last_interval,
                   int blocks_num, int threads_num)
{
  float *d_filtered;
  hipMalloc(&d_filtered, sizeof(float) * height_band * width_band);

  int indexes[1] = {0};
  int *d_indexes;
  hipMalloc((void **)&d_indexes, sizeof(int) * 1);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 1, hipMemcpyHostToDevice);

  filter_valid_values<<<blocks_num, threads_num>>>(d_target, d_filtered, height_band, width_band, d_indexes);

  hipMemcpy(&indexes[0], d_indexes, sizeof(int), hipMemcpyDeviceToHost);

  // Use Thrust to sort the valid elements on the GPU
  thrust::device_ptr<float> d_filtered_ptr = thrust::device_pointer_cast(d_filtered);
  thrust::sort(thrust::device, d_filtered_ptr, d_filtered_ptr + indexes[0]);

  int first_index = static_cast<int>(floor(first_interval * indexes[0]));
  int middle_index = static_cast<int>(floor(middle_interval * indexes[0]));
  int last_index = static_cast<int>(floor(last_interval * indexes[0]));

  hipMemcpy(&v_quartile[0], d_filtered + first_index, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&v_quartile[1], d_filtered + middle_index, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&v_quartile[2], d_filtered + last_index, sizeof(float), hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(d_filtered);
  hipFree(d_indexes);
}

pair<Candidate, Candidate> getEndmembersSTEPP(float *ndvi, float *d_ndvi, float *surface_temperature, float *d_surface_temperature, float *albedo, float *d_albedo,
                                              float *net_radiation, float *d_net_radiation, float *soil_heat, float *d_soil_heat,
                                              int blocks_num, int threads_num, int height_band, int width_band)
{
  const size_t MAXC = sizeof(Candidate) * height_band * width_band;

  float *d_ho;
  int *d_indexes;
  Candidate *d_hotCandidates, *d_coldCandidates;
  hipMalloc((void **)&d_ho, sizeof(float) * height_band * width_band);

  int indexes[2] = {0, 0};
  hipMalloc((void **)&d_indexes, sizeof(int) * 2);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

  hipError_t err;
  err = hipMalloc((void **)&d_hotCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_hotCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  err = hipMalloc((void **)&d_coldCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_coldCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  Candidate *hotCandidates, *coldCandidates;
  hotCandidates = (Candidate *)malloc(MAXC);
  coldCandidates = (Candidate *)malloc(MAXC);

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles_cuda(d_ndvi, ndviQuartile.data(), height_band, width_band, 0.15, 0.97, 0.97, blocks_num, threads_num);
  get_quartiles_cuda(d_albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);
  get_quartiles_cuda(d_surface_temperature, tsQuartile.data(), height_band, width_band, 0.20, 0.85, 0.97, blocks_num, threads_num);

  process_pixels<<<blocks_num, threads_num>>>(d_hotCandidates, d_coldCandidates, d_indexes,
                                              d_ndvi, d_surface_temperature, d_albedo, d_net_radiation, d_soil_heat, d_ho,
                                              ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2],
                                              albedoQuartile[0], albedoQuartile[1], albedoQuartile[2], height_band, width_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  hipMemcpy(&indexes, d_indexes, sizeof(int) * 2, hipMemcpyDeviceToHost);
  hipMemcpy(hotCandidates, d_hotCandidates, sizeof(Candidate) * indexes[0], hipMemcpyDeviceToHost);
  hipMemcpy(coldCandidates, d_coldCandidates, sizeof(Candidate) * indexes[1], hipMemcpyDeviceToHost);

  std::sort(hotCandidates, hotCandidates + indexes[0], compare_candidate_temperature);
  std::sort(coldCandidates, coldCandidates + indexes[1], compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

pair<Candidate, Candidate> getEndmembersASEBAL(float *ndvi, float *d_ndvi, float *surface_temperature, float *d_surface_temperature, float *albedo, float *d_albedo,
                                              float *net_radiation, float *d_net_radiation, float *soil_heat, float *d_soil_heat,
                                              int blocks_num, int threads_num, int height_band, int width_band)
{
  const size_t MAXC = sizeof(Candidate) * height_band * width_band;

  float *d_ho;
  int *d_indexes;
  Candidate *d_hotCandidates, *d_coldCandidates;
  hipMalloc((void **)&d_ho, sizeof(float) * height_band * width_band);

  int indexes[2] = {0, 0};
  hipMalloc((void **)&d_indexes, sizeof(int) * 2);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

  hipError_t err;
  err = hipMalloc((void **)&d_hotCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_hotCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  err = hipMalloc((void **)&d_coldCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_coldCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  Candidate *hotCandidates, *coldCandidates;
  hotCandidates = (Candidate *)malloc(MAXC);
  coldCandidates = (Candidate *)malloc(MAXC);

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles_cuda(d_ndvi, ndviQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75, blocks_num, threads_num);
  get_quartiles_cuda(d_albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);
  get_quartiles_cuda(d_surface_temperature, tsQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75, blocks_num, threads_num);

  process_pixels<<<blocks_num, threads_num>>>(d_hotCandidates, d_coldCandidates, d_indexes,
                                              d_ndvi, d_surface_temperature, d_albedo, d_net_radiation, d_soil_heat, d_ho,
                                              ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2],
                                              albedoQuartile[0], albedoQuartile[1], albedoQuartile[2], height_band, width_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  hipMemcpy(&indexes, d_indexes, sizeof(int) * 2, hipMemcpyDeviceToHost);
  hipMemcpy(hotCandidates, d_hotCandidates, sizeof(Candidate) * indexes[0], hipMemcpyDeviceToHost);
  hipMemcpy(coldCandidates, d_coldCandidates, sizeof(Candidate) * indexes[1], hipMemcpyDeviceToHost);

  std::sort(hotCandidates, hotCandidates + indexes[0], compare_candidate_temperature);
  std::sort(coldCandidates, coldCandidates + indexes[1], compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

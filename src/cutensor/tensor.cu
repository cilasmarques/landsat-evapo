#include "tensor.h"

Tensor::Tensor()
{
  HANDLE_CUTENSOR_ERROR(hiptensorCreate(&this->handle));
  HANDLE_ERROR(hipStreamCreate(&this->stream));
}

void Tensor::createPlanWork(hiptensorOperationDescriptor_t desc)
{
  // Optional (but recommended): ensure that the scalar type is correct.
  hiptensorDataType_t scalarType;
  HANDLE_CUTENSOR_ERROR(hiptensorOperationDescriptorGetAttribute(handle, desc, HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE, (void *)&scalarType, sizeof(scalarType)));
  assert(scalarType == HIPTENSOR_R_32F);

  // Set the algorithm to use
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  hiptensorPlanPreference_t planPrefContraction;
  HANDLE_CUTENSOR_ERROR(hiptensorCreatePlanPreference(handle, &planPrefContraction, algo, HIPTENSOR_JIT_MODE_NONE));

  // Query workspace estimate
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  HANDLE_CUTENSOR_ERROR(hiptensorEstimateWorkspaceSize(handle, desc, planPrefContraction, workspacePref, &workspaceSizeEstimate));

  // Create Contraction Plan
  HANDLE_CUTENSOR_ERROR(hiptensorCreatePlan(handle, &plan, desc, planPrefContraction, workspaceSizeEstimate));

  // Optional: query actually used workspace
  HANDLE_CUTENSOR_ERROR(hiptensorPlanGetAttribute(handle, plan, HIPTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize, sizeof(actualWorkspaceSize)));
  assert(actualWorkspaceSize <= workspaceSizeEstimate);

  // Define the workspace
  if (actualWorkspaceSize > 0)
  {
    HANDLE_ERROR(hipMalloc(&work, actualWorkspaceSize));
    assert(uintptr_t(work) % 128 == 0); // workspace must be aligned to 128 byte-boundary
  }
}

void Tensor::createNormalContraction(int height_band, int width_band)
{
  int dim_num = 2;
  std::vector<int> axis{'m', 'n'};
  std::vector<int64_t> axis_dim = {height_band, width_band};

  const uint32_t kAlignment = 128;

  // Define descriptors
  hiptensorTensorDescriptor_t descA;
  hiptensorTensorDescriptor_t descB;
  hiptensorTensorDescriptor_t descC;
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &descA, dim_num, axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &descB, dim_num, axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &descC, dim_num, axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));

  hiptensorOperationDescriptor_t descContraction;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_CUTENSOR_ERROR(hiptensorCreateContraction(this->handle,
                                                  &descContraction,
                                                  descA, axis.data(), /* unary operator A*/ HIPTENSOR_OP_IDENTITY,
                                                  descB, axis.data(), /* unary operator B*/ HIPTENSOR_OP_IDENTITY,
                                                  descC, axis.data(), /* unary operator C*/ HIPTENSOR_OP_IDENTITY,
                                                  descC, axis.data(),
                                                  descCompute));
  createPlanWork(descContraction);
}

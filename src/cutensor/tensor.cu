#include "tensor.cuh"

Tensor::Tensor() {};

Tensor::Tensor(int height_band, int width_band)
{
  HANDLE_CUTENSOR_ERROR(hiptensorCreate(&this->handle));
  HANDLE_ERROR(hipStreamCreate(&this->stream));
  this->dim_num = 2;
  this->axis = {'m', 'n'};
  this->axis_dim = {height_band, width_band};

  const uint32_t kAlignment = 128;

  // Define descriptors
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &this->descA, this->dim_num, this->axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &this->descB, this->dim_num, this->axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &this->descC, this->dim_num, this->axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));
  HANDLE_CUTENSOR_ERROR(hiptensorCreateTensorDescriptor(this->handle, &this->descD, this->dim_num, this->axis_dim.data(), NULL, HIPTENSOR_R_32F, kAlignment));

  // Create tensors
  // == Trinary
  createTrinary(this->tensor_plan_trinity_add_mult, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_ADD, HIPTENSOR_OP_MUL);
  createTrinary(this->tensor_plan_trinity_mult_add, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_MUL, HIPTENSOR_OP_ADD);

  // == Binary
  createBinary(this->tensor_plan_binary_max, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_MAX);
  createBinary(this->tensor_plan_binary_min, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_MIN);
  createBinary(this->tensor_plan_binary_rcp, HIPTENSOR_OP_RCP, HIPTENSOR_OP_RCP, HIPTENSOR_OP_MUL);
  createBinary(this->tensor_plan_binary_add, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_ADD);
  createBinary(this->tensor_plan_binary_mult, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_MUL);
  createBinary(this->tensor_plan_binary_div, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_RCP, HIPTENSOR_OP_MUL);
  createBinary(this->tensor_plan_binary_sqtr_add, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_SQRT, HIPTENSOR_OP_ADD);
  createBinary(this->tensor_plan_binary_log_mul, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_LOG, HIPTENSOR_OP_MUL);
  createBinary(this->tensor_plan_binary_exp_mul, HIPTENSOR_OP_IDENTITY, HIPTENSOR_OP_EXP, HIPTENSOR_OP_MUL);

  // == Permutation
  createPermutation(this->tensor_plan_permute_id, HIPTENSOR_OP_IDENTITY);
  createPermutation(this->tensor_plan_permute_exp, HIPTENSOR_OP_EXP);
  createPermutation(this->tensor_plan_permute_log, HIPTENSOR_OP_LOG);
}

void Tensor::createTrinary(hiptensorPlan_t &plan, hiptensorOperator_t OPA, hiptensorOperator_t OPB, hiptensorOperator_t OPC, hiptensorOperator_t OPAB, hiptensorOperator_t OPABC)
{
  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_CUTENSOR_ERROR(hiptensorCreateElementwiseTrinary(this->handle, &desc,
                                                         this->descA, this->axis.data(), OPA,
                                                         this->descB, this->axis.data(), OPB,
                                                         this->descC, this->axis.data(), OPC,
                                                         this->descD, this->axis.data(),
                                                         OPAB, OPABC,
                                                         descCompute));
  createPlan(plan, desc);
}

void Tensor::createBinary(hiptensorPlan_t &plan, hiptensorOperator_t OPA, hiptensorOperator_t OPB, hiptensorOperator_t OPAB)
{
  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_CUTENSOR_ERROR(hiptensorCreateElementwiseBinary(this->handle, &desc,
                                                        this->descA, this->axis.data(), OPA,
                                                        this->descB, this->axis.data(), OPB,
                                                        this->descC, this->axis.data(),
                                                        OPAB, descCompute));
  createPlan(plan, desc);
}

void Tensor::createPermutation(hiptensorPlan_t &plan, hiptensorOperator_t OPA)
{
  hiptensorOperationDescriptor_t desc;
  const hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
  HANDLE_CUTENSOR_ERROR(hiptensorCreatePermutation(this->handle, &desc,
                                                  this->descA, this->axis.data(), OPA,
                                                  this->descB, this->axis.data(),
                                                  descCompute));
  createPlan(plan, desc);
}

void Tensor::createPlan(hiptensorPlan_t &plan, hiptensorOperationDescriptor_t desc)
{
  // Optional (but recommended): ensure that the scalar type is correct.
  hiptensorDataType_t scalarType;
  HANDLE_CUTENSOR_ERROR(hiptensorOperationDescriptorGetAttribute(this->handle, desc, HIPTENSOR_OPERATION_DESCRIPTOR_SCALAR_TYPE, (void *)&scalarType, sizeof(scalarType)));
  assert(scalarType == HIPTENSOR_R_32F);

  // Set the algorithm to use
  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
  hiptensorPlanPreference_t planPref;
  HANDLE_CUTENSOR_ERROR(hiptensorCreatePlanPreference(this->handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

  // Query workspace estimate
  uint64_t workspaceSizeEstimate = 0;
  const hiptensorWorksizePreference_t workspacePref = HIPTENSOR_WORKSPACE_DEFAULT;
  HANDLE_CUTENSOR_ERROR(hiptensorEstimateWorkspaceSize(this->handle, desc, planPref, workspacePref, &workspaceSizeEstimate));

  // Create Plan
  HANDLE_CUTENSOR_ERROR(hiptensorCreatePlan(this->handle, &plan, desc, planPref, workspaceSizeEstimate));
}


#include "hip/hip_runtime.h"
#include "endmembers.h"
#include "filter.cuh"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

void get_quartiles_cuda(float *d_target, float *v_quartile, int height_band, int width_band,
                        float first_interval, float middle_interval, float last_interval,
                        int blocks_num, int threads_num)
{
  float *d_filtered;
  hipMalloc(&d_filtered, sizeof(float) * height_band * width_band);

  int indexes[1] = {0};
  int *d_indexes;
  hipMalloc((void **)&d_indexes, sizeof(int) * 1);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 1, hipMemcpyHostToDevice);

  filter_valid_values<<<blocks_num, threads_num>>>(d_target, d_filtered, height_band, width_band, d_indexes);

  hipMemcpy(&indexes[0], d_indexes, sizeof(int), hipMemcpyDeviceToHost);

  // Use Thrust to sort the valid elements on the GPU
  thrust::device_ptr<float> d_filtered_ptr = thrust::device_pointer_cast(d_filtered);
  thrust::sort(thrust::device, d_filtered_ptr, d_filtered_ptr + indexes[0]);

  int first_index = static_cast<int>(floor(first_interval * indexes[0]));
  int middle_index = static_cast<int>(floor(middle_interval * indexes[0]));
  int last_index = static_cast<int>(floor(last_interval * indexes[0]));

  hipMemcpy(&v_quartile[0], d_filtered + first_index, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&v_quartile[1], d_filtered + middle_index, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&v_quartile[2], d_filtered + last_index, sizeof(float), hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(d_filtered);
  hipFree(d_indexes);
}

string getEndmembersSTEEP(float *ndvi, float *d_ndvi, float *surface_temperature, float *d_surface_temperature, float *albedo, float *d_albedo,
                          float *net_radiation, float *d_net_radiation, float *soil_heat, float *d_soil_heat, int blocks_num, int threads_num,
                          Candidate &hot_pixel, Candidate &cold_pixel, int height_band, int width_band)
{
  string result = "";
  int64_t initial_time, final_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  Candidate *d_hotCandidates, *d_coldCandidates;
  const size_t MAXC = sizeof(Candidate) * height_band * width_band;

  hipError_t err;
  err = hipMalloc((void **)&d_hotCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_hotCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  err = hipMalloc((void **)&d_coldCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_coldCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  float *d_ho;
  hipMalloc((void **)&d_ho, sizeof(float) * height_band * width_band);

  int *d_indexes;
  int indexes[2] = {0, 0};
  hipMalloc((void **)&d_indexes, sizeof(int) * 2);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

  Candidate *hotCandidates, *coldCandidates;
  hotCandidates = (Candidate *)malloc(MAXC);
  coldCandidates = (Candidate *)malloc(MAXC);

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);

  try
  {
    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    get_quartiles_cuda(d_ndvi, ndviQuartile.data(), height_band, width_band, 0.15, 0.97, 0.97, blocks_num, threads_num);
    get_quartiles_cuda(d_albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);
    get_quartiles_cuda(d_surface_temperature, tsQuartile.data(), height_band, width_band, 0.20, 0.85, 0.97, blocks_num, threads_num);

    process_pixels_STEEP<<<blocks_num, threads_num>>>(d_hotCandidates, d_coldCandidates, d_indexes,
                                                      d_ndvi, d_surface_temperature, d_albedo, d_net_radiation, d_soil_heat, d_ho,
                                                      ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2],
                                                      albedoQuartile[0], albedoQuartile[1], albedoQuartile[2], height_band, width_band);
    hipEventRecord(stop);

    hipMemcpy(&indexes, d_indexes, sizeof(int) * 2, hipMemcpyDeviceToHost);
    hipMemcpy(hotCandidates, d_hotCandidates, sizeof(Candidate) * indexes[0], hipMemcpyDeviceToHost);
    hipMemcpy(coldCandidates, d_coldCandidates, sizeof(Candidate) * indexes[1], hipMemcpyDeviceToHost);

    std::sort(hotCandidates, hotCandidates + indexes[0], compare_candidate_temperature);
    std::sort(coldCandidates, coldCandidates + indexes[1], compare_candidate_temperature);

    unsigned int hotPos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
    unsigned int coldPos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

    hot_pixel = hotCandidates[hotPos];
    cold_pixel = coldCandidates[coldPos];

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    result += "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
  }
  catch (const std::exception &e)
  {
    cerr << "Pixel filtering error: " << e.what() << endl;
    exit(15);
  }

  return result;
}

string getEndmembersASEBAL(float *ndvi, float *d_ndvi, float *surface_temperature, float *d_surface_temperature, float *albedo, float *d_albedo,
                           float *net_radiation, float *d_net_radiation, float *soil_heat, float *d_soil_heat, int blocks_num, int threads_num,
                           Candidate &hot_pixel, Candidate &cold_pixel, int height_band, int width_band)
{
  string result = "";
  int64_t initial_time, final_time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  Candidate *d_hotCandidates, *d_coldCandidates;
  const size_t MAXC = sizeof(Candidate) * height_band * width_band;

  hipError_t err;
  err = hipMalloc((void **)&d_hotCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_hotCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  err = hipMalloc((void **)&d_coldCandidates, MAXC);
  if (err != hipSuccess)
  {
    std::cerr << "CUDA memory allocation for d_coldCandidates failed: " << hipGetErrorString(err) << std::endl;
    // Handle the error appropriately
  }

  float *d_ho;
  hipMalloc((void **)&d_ho, sizeof(float) * height_band * width_band);

  int *d_indexes;
  int indexes[2] = {0, 0};
  hipMalloc((void **)&d_indexes, sizeof(int) * 2);
  hipMemcpy(d_indexes, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

  Candidate *hotCandidates, *coldCandidates;
  hotCandidates = (Candidate *)malloc(MAXC);
  coldCandidates = (Candidate *)malloc(MAXC);

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);

  try
  {
    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    get_quartiles_cuda(d_ndvi, ndviQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);
    get_quartiles_cuda(d_albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);
    get_quartiles_cuda(d_surface_temperature, tsQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75, blocks_num, threads_num);

    process_pixels_ASEBAL<<<blocks_num, threads_num>>>(d_hotCandidates, d_coldCandidates, d_indexes,
                                                       d_ndvi, d_surface_temperature, d_albedo, d_net_radiation, d_soil_heat, d_ho,
                                                       ndviQuartile[0], ndviQuartile[2], tsQuartile[1], tsQuartile[0],
                                                       albedoQuartile[1], albedoQuartile[1], height_band, width_band);
    hipEventRecord(stop);

    hipMemcpy(&indexes, d_indexes, sizeof(int) * 2, hipMemcpyDeviceToHost);
    hipMemcpy(hotCandidates, d_hotCandidates, sizeof(Candidate) * indexes[0], hipMemcpyDeviceToHost);
    hipMemcpy(coldCandidates, d_coldCandidates, sizeof(Candidate) * indexes[1], hipMemcpyDeviceToHost);

    std::sort(hotCandidates, hotCandidates + indexes[0], compare_candidate_temperature);
    std::sort(coldCandidates, coldCandidates + indexes[1], compare_candidate_temperature);

    unsigned int hotPos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
    unsigned int coldPos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

    hot_pixel = hotCandidates[hotPos];
    cold_pixel = coldCandidates[coldPos];

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    result += "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
  }
  catch (const std::exception &e)
  {
    cerr << "Pixel filtering error: " << e.what() << endl;
    exit(15);
  }

  return result;
}

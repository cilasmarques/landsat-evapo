#include "hip/hip_runtime.h"
#include "kernels.cuh"

__device__ int width_d;
__device__ int height_d;

__global__ void rad_kernel(float *band_d, float *radiance_d, float *rad_add_d, float *rad_mult_d, int band_idx)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        radiance_d[pos] = band_d[pos] * rad_mult_d[band_idx] + rad_add_d[band_idx];

        if (radiance_d[pos] <= 0)
            radiance_d[pos] = NAN;
    }
}

__global__ void ref_kernel(float *band_d, float *reflectance_d, float *ref_add_d, float *ref_mult_d, float sin_sun, int band_idx)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        reflectance_d[pos] = (band_d[pos] * ref_mult_d[band_idx] + ref_add_d[band_idx]) / sin_sun;

        if (reflectance_d[pos] <= 0)
            reflectance_d[pos] = NAN;
    }
}

__global__ void albedo_kernel(float *reflectance_blue_d, float *reflectance_green_d, float *reflectance_red_d, float *reflectance_nir_d, float *reflectance_swir1_d, float *reflectance_swir2_d, float *tal_d, float *albedo_d, float *ref_w_coeff_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float alb_toa = reflectance_blue_d[pos] * ref_w_coeff_d[PARAM_BAND_BLUE_INDEX] +
                        reflectance_green_d[pos] * ref_w_coeff_d[PARAM_BAND_GREEN_INDEX] +
                        reflectance_red_d[pos] * ref_w_coeff_d[PARAM_BAND_RED_INDEX] +
                        reflectance_nir_d[pos] * ref_w_coeff_d[PARAM_BAND_NIR_INDEX] +
                        reflectance_swir1_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR1_INDEX] +
                        reflectance_swir2_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR2_INDEX];

        albedo_d[pos] = (alb_toa - 0.03f) / (tal_d[pos] * tal_d[pos]);

        if (albedo_d[pos] <= 0)
            albedo_d[pos] = NAN;
    }
}

__global__ void ndvi_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *ndvi_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ndvi_d[pos] = (reflectance_nir_d[pos] - reflectance_red_d[pos]) / (reflectance_nir_d[pos] + reflectance_red_d[pos]);

        if (ndvi_d[pos] <= -1 || ndvi_d[pos] >= 1)
            ndvi_d[pos] = NAN;
    }
}

__global__ void pai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *pai_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        pai_d[pos] = 10.1f * (reflectance_nir_d[pos] - sqrtf(reflectance_red_d[pos])) + 3.1f;

        if (pai_d[pos] < 0)
            pai_d[pos] = 0;
    }
}

__global__ void lai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *lai_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float savi = ((1.5f) * (reflectance_nir_d[pos] - reflectance_red_d[pos])) / (0.5f + (reflectance_nir_d[pos] + reflectance_red_d[pos]));

        if (!isnan(savi) && savi > 0.687f)
            lai_d[pos] = 6;
        if (!isnan(savi) && savi <= 0.687f)
            lai_d[pos] = -logf((0.69f - savi) / 0.59f) / 0.91f;
        if (!isnan(savi) && savi < 0.1)
            lai_d[pos] = 0;

        if (lai_d[pos] < 0)
            lai_d[pos] = 0;
    }
}

__global__ void enb_kernel(float *lai_d, float *ndvi_d, float *enb_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        if (ndvi_d[pos] > 0)
            enb_d[pos] = (lai_d[pos] < 3) ? 0.97f + 0.0033f * lai_d[pos] : 0.98f;            
        else if (ndvi_d[pos] < 0)
            enb_d[pos] = 0.99;
        else
            enb_d[pos] = NAN;
    }
}

__global__ void eo_kernel(float *lai_d, float *ndvi_d, float *eo_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        if (ndvi_d[pos] > 0)
            eo_d[pos] = (lai_d[pos] < 3) ? 0.95f + 0.01f * lai_d[pos] : 0.98f;
        else if (ndvi_d[pos] < 0)
            eo_d[pos] = 0.985;
        else
            eo_d[pos] = NAN;
    }
}

__global__ void ea_kernel(float *tal_d, float *ea_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ea_d[pos] = 0.85f * powf((-1 * logf(tal_d[pos])), 0.09f);
    }
}

__global__ void surface_temperature_kernel(float *enb_d, float *radiance_termal_d, float *surface_temperature_d, float k1, float k2)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        surface_temperature_d[pos] = k2 / (logf((enb_d[pos] * k1 / radiance_termal_d[pos]) + 1));

        if (surface_temperature_d[pos] < 0)
            surface_temperature_d[pos] = 0;
    }
}

__global__ void short_wave_radiation_kernel(float *tal_d, float *short_wave_radiation_d, float sun_elevation, float distance_earth_sun, float pi)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        short_wave_radiation_d[pos] = (1367.0f * sinf(sun_elevation * pi / 180.0f) * tal_d[pos]) / (distance_earth_sun * distance_earth_sun);
    }
}

__global__ void large_waves_radiation_kernel(float *surface_temperature_d, float *eo_d, float *ea_d, float *large_wave_radiation_atmosphere_d, float *large_wave_radiation_surface_d, float temperature)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float temperature_pixel = surface_temperature_d[pos];
        float surface_temperature_pow_4 = temperature_pixel * temperature_pixel * temperature_pixel * temperature_pixel;
        large_wave_radiation_surface_d[pos] = eo_d[pos] * 5.67f * 1e-8f * surface_temperature_pow_4;
        
        float station_temperature_kelvin_pow_4 = temperature * temperature * temperature * temperature;
        large_wave_radiation_atmosphere_d[pos] = ea_d[pos] * 5.67f * 1e-8f * station_temperature_kelvin_pow_4;
    }
}

__global__ void net_radiation_kernel(float *short_wave_radiation_d, float *albedo_d, float *large_wave_radiation_atmosphere_d, float *large_wave_radiation_surface_d, float *eo_d, float *net_radiation_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        net_radiation_d[pos] = (1 - albedo_d[pos]) * short_wave_radiation_d[pos] + large_wave_radiation_atmosphere_d[pos] - large_wave_radiation_surface_d[pos] - (1 - eo_d[pos]) * large_wave_radiation_atmosphere_d[pos];

        if (net_radiation_d[pos] < 0)
            net_radiation_d[pos] = 0;
    }
}

__global__ void soil_heat_kernel(float *ndvi_d, float *albedo_d, float *surface_temperature_d, float *net_radiation_d, float *soil_heat_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        if (ndvi_d[pos] >= 0) {
            float temperature_celcius = surface_temperature_d[pos] - 273.15f;
            float ndvi_pixel_pow_4 = ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos];
            soil_heat_d[pos] = temperature_celcius * (0.0038f + 0.0074f * albedo_d[pos]) * (1.0f - 0.98f * ndvi_pixel_pow_4) * net_radiation_d[pos];
        } else            
            soil_heat_d[pos] = 0.5f * net_radiation_d[pos];

        if (soil_heat_d[pos] < 0)
            soil_heat_d[pos] = 0;
    }
}

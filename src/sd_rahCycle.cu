#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "kernels.cuh"
#include "sensors.cuh"
#include "surfaceData.cuh"

string d0_fuction(Products products)
{
    float CD1 = 20.6;
    float HGHT = 4;

    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    d0_kernel<<<blocks_n, threads_n>>>(products.pai_d, products.d0_d, CD1, HGHT);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,D0," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string kb_function(Products products, float ndvi_max, float ndvi_min)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    kb_kernel<<<blocks_n, threads_n>>>(products.zom_d, products.ustar_d, products.pai_d, products.kb1_d, products.ndvi_d, ndvi_max, ndvi_min);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,KB1," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string zom_fuction(Products products, float A_ZOM, float B_ZOM)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    if (model_method == 0)
        zom_kernel_STEEP<<<blocks_n, threads_n>>>(products.d0_d, products.pai_d, products.zom_d, A_ZOM, B_ZOM);
    else
        zom_kernel_ASEBAL<<<blocks_n, threads_n>>>(products.ndvi_d, products.albedo_d, products.zom_d, A_ZOM, B_ZOM);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,ZOM," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string ustar_fuction(Products products, float u_const)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    if (model_method == 0)
        ustar_kernel_STEEP<<<blocks_n, threads_n>>>(products.zom_d, products.d0_d, products.ustar_d, u_const);
    else
        ustar_kernel_ASEBAL<<<blocks_n, threads_n>>>(products.zom_d, products.ustar_d, u_const);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,USTAR," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string aerodynamic_resistance_fuction(Products products)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    if (model_method == 0)
        aerodynamic_resistance_kernel_STEEP<<<blocks_n, threads_n>>>(products.zom_d, products.d0_d, products.ustar_d, products.kb1_d, products.rah_d);
    else
        aerodynamic_resistance_kernel_ASEBAL<<<blocks_n, threads_n>>>(products.ustar_d, products.rah_d);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,RAH_INI," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string rah_correction_function_blocks_STEEP(Products products, float ndvi_min, float ndvi_max)
{
    string result = "";
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int64_t initial_time, final_time;

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    // ========= CUDA Setup
    int dev = 0;
    hipDeviceProp_t deviceProp;
    HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
    HANDLE_ERROR(hipSetDevice(dev));

    hipEventRecord(start);
    for (int i = 0; i < 2; i++) {
        rah_correction_cycle_STEEP<<<blocks_n, threads_n>>>(products.net_radiation_d, products.soil_heat_d, products.ndvi_d, products.surface_temperature_d, products.d0_d, products.kb1_d, products.zom_d, products.ustar_d, products.rah_d, products.sensible_heat_flux_d, ndvi_max, ndvi_min);
    }
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    return "KERNELS,RAH_CYCLE," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}

string rah_correction_function_blocks_ASEBAL(Products products, float u200)
{
    string result = "";
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int64_t initial_time, final_time;

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    // ========= CUDA Setup
    int dev = 0;
    hipDeviceProp_t deviceProp;
    HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
    HANDLE_ERROR(hipSetDevice(dev));

    hipEventRecord(start);
    int i = 0;
    while (true) {
        rah_correction_cycle_ASEBAL<<<blocks_n, threads_n>>>(products.net_radiation_d, products.soil_heat_d, products.ndvi_d, products.surface_temperature_d, products.kb1_d, products.zom_d, products.ustar_d, products.rah_d, products.sensible_heat_flux_d, u200, products.stop_condition_d);

        HANDLE_ERROR(hipMemcpy(products.stop_condition, products.stop_condition_d, sizeof(int), hipMemcpyDeviceToHost));

        if (i > 0 && *products.stop_condition)
            break;
        else
            i++;
    }
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    return "KERNELS,RAH_CYCLE," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}


string sensible_heat_flux_function(Products products)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    sensible_heat_flux_kernel<<<blocks_n, threads_n>>>(products.surface_temperature_d, products.rah_d, products.net_radiation_d, products.soil_heat_d, products.sensible_heat_flux_d);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,SENSIBLE_HEAT_FLUX," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string Products::converge_rah_cycle(Products products, Station station)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    float ustar_station = (VON_KARMAN * station.v6) / (logf(station.WIND_SPEED / station.SURFACE_ROUGHNESS));
    float u10 = (ustar_station / VON_KARMAN) * logf(10.0f / station.SURFACE_ROUGHNESS);
    float u200 = (ustar_station / VON_KARMAN) * logf(200.0f / station.SURFACE_ROUGHNESS);

    thrust::device_ptr<float> ndvi_ptr = thrust::device_pointer_cast(products.ndvi_d);
    
    float ndvi_min = thrust::reduce(ndvi_ptr, 
                                   ndvi_ptr + products.height_band * products.width_band,
                                   1.0f, // Initial value
                                   min_valid());
    
    float ndvi_max = thrust::reduce(ndvi_ptr, 
                                   ndvi_ptr + products.height_band * products.width_band,
                                   -1.0f, // Initial value
                                   max_valid());

    if (model_method == 0) { // STEEP
        result += d0_fuction(products);
        result += zom_fuction(products, station.A_ZOM, station.B_ZOM);
        result += ustar_fuction(products, u10);
        result += kb_function(products, ndvi_max, ndvi_min);
        result += aerodynamic_resistance_fuction(products);
        result += rah_correction_function_blocks_STEEP(products, ndvi_min, ndvi_max);
        result += sensible_heat_flux_function(products);
    } else { // ASEBAL
        result += zom_fuction(products, station.A_ZOM, station.B_ZOM);
        result += ustar_fuction(products, u200);
        result += aerodynamic_resistance_fuction(products);
        result += rah_correction_function_blocks_ASEBAL(products, u200);
        result += sensible_heat_flux_function(products);
    }
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    result += "KERNELS,P3_RAH," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
    return result;
};

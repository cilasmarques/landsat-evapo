#include "hip/hip_runtime.h"
#include "constants.h"
#include "kernels.cuh"
#include "surfaceData.cuh"

__host__ __device__ Endmember::Endmember()
{
    this->ndvi = 0;
    this->temperature = 0;
    this->line = 0;
    this->col = 0;
}

__host__ __device__ Endmember::Endmember(double ndvi, double temperature, int line, int col)
{
    this->ndvi = ndvi;
    this->temperature = temperature;
    this->line = line;
    this->col = col;
}

void get_quartiles_cuda(double *d_target, double *v_quartile, int height_band, int width_band, double first_interval, double middle_interval, double last_interval, int blocks_n, int threads_n)
{
    double *d_filtered;
    hipMalloc(&d_filtered, sizeof(double) * height_band * width_band);

    int indexes[1] = {0};
    int *indexes_d;
    hipMalloc((void **)&indexes_d, sizeof(int) * 1);
    hipMemcpy(indexes_d, indexes, sizeof(int) * 1, hipMemcpyHostToDevice);

    filter_valid_values<<<blocks_n, threads_n>>>(d_target, d_filtered, indexes_d);

    hipMemcpy(&indexes[0], indexes_d, sizeof(int), hipMemcpyDeviceToHost);

    // Use Thrust to sort the valid elements on the GPU
    thrust::device_ptr<double> d_filtered_ptr = thrust::device_pointer_cast(d_filtered);
    thrust::sort(thrust::device, d_filtered_ptr, d_filtered_ptr + indexes[0]);

    int first_index = static_cast<int>(floor(first_interval * indexes[0]));
    int middle_index = static_cast<int>(floor(middle_interval * indexes[0]));
    int last_index = static_cast<int>(floor(last_interval * indexes[0]));

    double temp_value;
    hipMemcpy(&temp_value, d_filtered + first_index, sizeof(double), hipMemcpyDeviceToHost);
    v_quartile[0] = (double)temp_value;
    hipMemcpy(&temp_value, d_filtered + middle_index, sizeof(double), hipMemcpyDeviceToHost);
    v_quartile[1] = (double)temp_value;
    hipMemcpy(&temp_value, d_filtered + last_index, sizeof(double), hipMemcpyDeviceToHost);
    v_quartile[2] = (double)temp_value;

    // Free GPU memory
    hipFree(d_filtered);
    hipFree(indexes_d);
}

string getEndmembersSTEEP(Products products, int *indexes_d)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    vector<double> tsQuartile(3);
    vector<double> ndviQuartile(3);
    vector<double> albedoQuartile(3);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    get_quartiles_cuda(products.ndvi_d, ndviQuartile.data(), products.height_band, products.width_band, 0.15, 0.97, 0.97, blocks_n, threads_n);
    get_quartiles_cuda(products.albedo_d, albedoQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
    get_quartiles_cuda(products.surface_temperature_d, tsQuartile.data(), products.height_band, products.width_band, 0.20, 0.85, 0.97, blocks_n, threads_n);

    process_pixels_STEEP<<<blocks_n, threads_n>>>(products.hotCandidates_d, products.coldCandidates_d, indexes_d, products.ndvi_d, products.surface_temperature_d, products.albedo_d, products.net_radiation_d, products.soil_heat_d, ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2], albedoQuartile[0], albedoQuartile[1], albedoQuartile[2]);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    return "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}

string getEndmembersASEBAL(Products products, int *indexes_d)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    vector<double> tsQuartile(3);
    vector<double> ndviQuartile(3);
    vector<double> albedoQuartile(3);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    get_quartiles_cuda(products.ndvi_d, ndviQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
    get_quartiles_cuda(products.albedo_d, albedoQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
    get_quartiles_cuda(products.surface_temperature_d, tsQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);

    process_pixels_ASEBAL<<<blocks_n, threads_n>>>(products.hotCandidates_d, products.coldCandidates_d, indexes_d, products.ndvi_d, products.surface_temperature_d, products.albedo_d, products.net_radiation_d, products.soil_heat_d, ndviQuartile[0], ndviQuartile[2], tsQuartile[2], tsQuartile[0], albedoQuartile[2], albedoQuartile[1]);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    return "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
}

string Products::select_endmembers(Products products)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    try
    {
        initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
        hipEventRecord(start);

        int *indexes_d;
        int indexes[2] = {0, 0};
        hipMalloc((void **)&indexes_d, sizeof(int) * 2);
        hipMemcpy(indexes_d, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

        if (model_method == 0)
            result += getEndmembersSTEEP(products, indexes_d);
        else if (model_method == 1)
            result += getEndmembersASEBAL(products, indexes_d);

        hipMemcpy(&indexes, indexes_d, sizeof(int) * 2, hipMemcpyDeviceToHost);
        int hot_pos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
        int cold_pos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

        if (indexes[0] == 0)
            throw std::runtime_error("No hot candidates found");
        if (indexes[1] == 0)
            throw std::runtime_error("No cold candidates found");

        // The dev_ptr_hot sort also sorts the hotCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_hot(products.hotCandidates_d);
        thrust::sort(dev_ptr_hot, dev_ptr_hot + indexes[0], CompareEndmemberTemperature());

        // The dev_ptr_cold sort also sorts the coldCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_cold(products.coldCandidates_d);
        thrust::sort(dev_ptr_cold, dev_ptr_cold + indexes[1], CompareEndmemberTemperature());

        Endmember hotCandidate = Endmember();
        Endmember coldCandidate = Endmember();

        hipMemcpy(&hotCandidate, products.hotCandidates_d + hot_pos, sizeof(Endmember), hipMemcpyDeviceToHost);
        hipMemcpy(&coldCandidate, products.coldCandidates_d + cold_pos, sizeof(Endmember), hipMemcpyDeviceToHost);

        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberLine_d), &hotCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberCol_d), &hotCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberLine_d), &coldCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberCol_d), &coldCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);
        hipEventRecord(stop);

        float cuda_time = 0;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cuda_time, start, stop);
        final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
        result += "KERNELS,P2_PIXEL_SEL," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
        return result;
    }
    catch (const std::exception &e)
    {
        cerr << "Pixel filtering error: " << e.what() << endl;
        exit(15);
    }

    return result;
}

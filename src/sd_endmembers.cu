#include "hip/hip_runtime.h"
#include "constants.h"
#include "kernels.cuh"
#include "surfaceData.cuh"

__host__ __device__ Endmember::Endmember()
{
    this->ndvi = 0;
    this->temperature = 0;
    this->line = 0;
    this->col = 0;
}

__host__ __device__ Endmember::Endmember(float ndvi, float temperature, int line, int col)
{
    this->ndvi = ndvi;
    this->temperature = temperature;
    this->line = line;
    this->col = col;
}

void get_quartiles_cuda(float *d_target, float *v_quartile, int height_band, int width_band, float first_interval, float middle_interval, float last_interval, int blocks_n, int threads_n)
{
    float *d_filtered;
    hipMalloc(&d_filtered, sizeof(float) * height_band * width_band);

    int indexes[1] = {0};
    int *indexes_d;
    hipMalloc((void **)&indexes_d, sizeof(int) * 1);
    hipMemcpy(indexes_d, indexes, sizeof(int) * 1, hipMemcpyHostToDevice);

    filter_valid_values<<<blocks_n, threads_n>>>(d_target, d_filtered, indexes_d);

    hipMemcpy(&indexes[0], indexes_d, sizeof(int), hipMemcpyDeviceToHost);

    // Use Thrust to sort the valid elements on the GPU
    thrust::device_ptr<float> d_filtered_ptr = thrust::device_pointer_cast(d_filtered);
    thrust::sort(thrust::device, d_filtered_ptr, d_filtered_ptr + indexes[0]);

    int first_index = static_cast<int>(floor(first_interval * indexes[0]));
    int middle_index = static_cast<int>(floor(middle_interval * indexes[0]));
    int last_index = static_cast<int>(floor(last_interval * indexes[0]));

    hipMemcpy(&v_quartile[0], d_filtered + first_index, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&v_quartile[1], d_filtered + middle_index, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&v_quartile[2], d_filtered + last_index, sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_filtered);
    hipFree(indexes_d);
}

string getEndmembersSTEEP(Products products)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *indexes_d;
    int indexes[2] = {0, 0};
    hipMalloc((void **)&indexes_d, sizeof(int) * 2);
    hipMemcpy(indexes_d, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

    vector<float> tsQuartile(3);
    vector<float> ndviQuartile(3);
    vector<float> albedoQuartile(3);

    try {
        initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

        hipEventRecord(start);
        get_quartiles_cuda(products.ndvi_d, ndviQuartile.data(), products.height_band, products.width_band, 0.15, 0.97, 0.97, blocks_n, threads_n);
        get_quartiles_cuda(products.albedo_d, albedoQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
        get_quartiles_cuda(products.surface_temperature_d, tsQuartile.data(), products.height_band, products.width_band, 0.20, 0.85, 0.97, blocks_n, threads_n);

        process_pixels_STEEP<<<blocks_n, threads_n>>>(products.hotCandidates_d, products.coldCandidates_d, indexes_d, products.ndvi_d, products.surface_temperature_d, products.albedo_d, products.net_radiation_d, products.soil_heat_d, ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2], albedoQuartile[0], albedoQuartile[1], albedoQuartile[2]);
        hipEventRecord(stop);

        hipMemcpy(&indexes, indexes_d, sizeof(int) * 2, hipMemcpyDeviceToHost);
        int hot_pos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
        int cold_pos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

        if (indexes[0] == 0)
            throw std::runtime_error("No hot candidates found");
        if (indexes[1] == 0)
            throw std::runtime_error("No cold candidates found");

        // The dev_ptr_hot sort also sorts the hotCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_hot(products.hotCandidates_d);
        thrust::sort(dev_ptr_hot, dev_ptr_hot + indexes[0], CompareEndmemberTemperature());

        // The dev_ptr_cold sort also sorts the coldCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_cold(products.coldCandidates_d);
        thrust::sort(dev_ptr_cold, dev_ptr_cold + indexes[1], CompareEndmemberTemperature());

        Endmember hotCandidate = Endmember();
        Endmember coldCandidate = Endmember();

        hipMemcpy(&hotCandidate, products.hotCandidates_d + hot_pos, sizeof(Endmember), hipMemcpyDeviceToHost);
        hipMemcpy(&coldCandidate, products.coldCandidates_d + cold_pos, sizeof(Endmember), hipMemcpyDeviceToHost);

        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberLine_d), &hotCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberCol_d), &hotCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberLine_d), &coldCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberCol_d), &coldCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);

        float cuda_time = 0;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cuda_time, start, stop);
        final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
        result += "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
    } catch (const std::exception &e) {
        cerr << "Pixel filtering error: " << e.what() << endl;
        exit(15);
    }

    return result;
}

string getEndmembersASEBAL(Products products)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *indexes_d;
    int indexes[2] = {0, 0};
    hipMalloc((void **)&indexes_d, sizeof(int) * 2);
    hipMemcpy(indexes_d, indexes, sizeof(int) * 2, hipMemcpyHostToDevice);

    vector<float> tsQuartile(3);
    vector<float> ndviQuartile(3);
    vector<float> albedoQuartile(3);

    try {
        initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

        hipEventRecord(start);
        get_quartiles_cuda(products.ndvi_d, ndviQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
        get_quartiles_cuda(products.albedo_d, albedoQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);
        get_quartiles_cuda(products.surface_temperature_d, tsQuartile.data(), products.height_band, products.width_band, 0.25, 0.50, 0.75, blocks_n, threads_n);

        process_pixels_ASEBAL<<<blocks_n, threads_n>>>(products.hotCandidates_d, products.coldCandidates_d, indexes_d, products.ndvi_d, products.surface_temperature_d, products.albedo_d, products.net_radiation_d, products.soil_heat_d, ndviQuartile[0], ndviQuartile[2], tsQuartile[2], tsQuartile[0], albedoQuartile[2], albedoQuartile[1]);
        hipEventRecord(stop);

        hipMemcpy(&indexes, indexes_d, sizeof(int) * 2, hipMemcpyDeviceToHost);
        int hot_pos = static_cast<unsigned int>(std::floor(indexes[0] * 0.5));
        int cold_pos = static_cast<unsigned int>(std::floor(indexes[1] * 0.5));

        if (indexes[0] == 0)
            throw std::runtime_error("No hot candidates found");
        if (indexes[1] == 0)
            throw std::runtime_error("No cold candidates found");

        // The dev_ptr_hot sort also sorts the hotCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_hot(products.hotCandidates_d);
        thrust::sort(dev_ptr_hot, dev_ptr_hot + indexes[0], CompareEndmemberTemperature());

        // The dev_ptr_cold sort also sorts the coldCandidates_d array
        thrust::device_ptr<Endmember> dev_ptr_cold(products.coldCandidates_d);
        thrust::sort(dev_ptr_cold, dev_ptr_cold + indexes[1], CompareEndmemberTemperature());

        Endmember hotCandidate = Endmember();
        Endmember coldCandidate = Endmember();

        hipMemcpy(&hotCandidate, products.hotCandidates_d + hot_pos, sizeof(Endmember), hipMemcpyDeviceToHost);
        hipMemcpy(&coldCandidate, products.coldCandidates_d + cold_pos, sizeof(Endmember), hipMemcpyDeviceToHost);

        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberLine_d), &hotCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(hotEndmemberCol_d), &hotCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberLine_d), &coldCandidate.line, sizeof(int), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(coldEndmemberCol_d), &coldCandidate.col, sizeof(int), 0, hipMemcpyHostToDevice);
        
        float cuda_time = 0;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&cuda_time, start, stop);
        final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
        result += "KERNELS,PIXEL_FILTER," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
    } catch (const std::exception &e) {
        cerr << "Pixel filtering error: " << e.what() << endl;
        exit(15);
    }

    return result;
}

string Products::select_endmembers(Products products)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    if (model_method == 0) { // STEEP
        result += getEndmembersSTEEP(products);
    } else if (model_method == 1) { // ASEBAL
        result += getEndmembersASEBAL(products);
    }
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    result += "KERNELS,P2_PIXEL_SEL," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
    return result;
}

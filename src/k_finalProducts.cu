#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void latent_heat_flux_kernel(float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d, float *latent_heat_flux_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        latent_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos] - sensible_heat_flux_d[pos];
    }
}

__global__ void net_radiation_24h_kernel(float *albedo_d, float Rs24h, float Ra24h, float *net_radiation_24h_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;
    float FL = 110.0;
    if (pos < width_d * height_d) {
        net_radiation_24h_d[pos] = (1.0 - albedo_d[pos]) * Rs24h - FL * Rs24h / Ra24h;
    }
}

__global__ void evapotranspiration_24h_kernel(float *surface_temperature_d, float *latent_heat_flux_d, float *net_radiation_d, float *soil_heat_d, float *net_radiation_24h_d, float *evapotranspiration_24h_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float temperature_celcius = surface_temperature_d[pos] - 273.15;
        evapotranspiration_24h_d[pos] = (86400.0 / ((2.501 - 0.0236 * temperature_celcius) * pow(10.0, 6.0))) * 
                                        (latent_heat_flux_d[pos] / (net_radiation_d[pos] - soil_heat_d[pos])) * 
                                        net_radiation_24h_d[pos];
    }
}

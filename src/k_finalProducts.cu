#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void latent_heat_flux_kernel(double *net_radiation_d, double *soil_heat_d, double *sensible_heat_flux_d, double *latent_heat_flux_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        latent_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos] - sensible_heat_flux_d[pos];
    }
}

__global__ void net_radiation_24h_kernel(double *albedo_d, float Rs24h, float Ra24h, double *net_radiation_24h_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;
    float FL = 110.0f;
    if (pos < width_d * height_d) {
        net_radiation_24h_d[pos] = (1.0f - albedo_d[pos]) * Rs24h - FL * Rs24h / Ra24h;
    }
}

__global__ void evapotranspiration_24h_kernel(double *surface_temperature_d, double *latent_heat_flux_d, double *net_radiation_d, double *soil_heat_d, double *net_radiation_24h_d, double *evapotranspiration_24h_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float temperature_celcius = surface_temperature_d[pos] - 273.15f;
        evapotranspiration_24h_d[pos] = (86400.0f / ((2.501f - 0.0236f * temperature_celcius) * powf(10.0f, 6.0f))) * 
                                        (latent_heat_flux_d[pos] / (net_radiation_d[pos] - soil_heat_d[pos])) * 
                                        net_radiation_24h_d[pos];
    }
}

#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void filter_valid_values(const float *target, float *filtered, int *ipos)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < height_d * width_d) {
        float value = target[pos];
        if (!isnan(value) && !isinf(value)) {
            int position = atomicAdd(ipos, 1);
            filtered[position] = value;
        }
    }
}

__global__ void process_pixels_STEEP(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surface_temperature_d, float *albedo_d, float ndviQuartileLow, float ndviQuartileHigh, float tsQuartileLow, float tsQuartileMid, float tsQuartileHigh, float albedoQuartileLow, float albedoQuartileMid, float albedoQuartileHigh)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10f && ndvi_d[pos] < ndviQuartileLow;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileMid && albedo_d[pos] < albedoQuartileHigh;
        bool hotTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] > tsQuartileMid && surface_temperature_d[pos] < tsQuartileHigh;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviQuartileHigh;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileLow && albedo_d[pos] < albedoQuartileMid;
        bool coldTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] < tsQuartileLow;

        if (hotAlbedo && hotNDVI && hotTS) {
            unsigned int row = pos / width_d;
            unsigned int col = pos % width_d;
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = {static_cast<uint16_t>(row), static_cast<uint16_t>(col), albedo_d[pos], ndvi_d[pos], surface_temperature_d[pos]};
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            unsigned int row = pos / width_d;
            unsigned int col = pos % width_d;
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = {static_cast<uint16_t>(row), static_cast<uint16_t>(col), albedo_d[pos], ndvi_d[pos], surface_temperature_d[pos]};
        }
    }
}

__global__ void process_pixels_ASEBAL(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surface_temperature_d, float *albedo_d, float ndviHOTQuartile, float ndviCOLDQuartile, float tsHOTQuartile, float tsCOLDQuartile, float albedoHOTQuartile, float albedoCOLDQuartile)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10f && ndvi_d[pos] < ndviHOTQuartile;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoHOTQuartile;
        bool hotTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] > tsHOTQuartile;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviCOLDQuartile;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] < albedoCOLDQuartile;
        bool coldTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] < tsCOLDQuartile;

        if (hotAlbedo && hotNDVI && hotTS) {
            unsigned int row = pos / width_d;
            unsigned int col = pos % width_d;
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = {static_cast<uint16_t>(row), static_cast<uint16_t>(col), albedo_d[pos], ndvi_d[pos], surface_temperature_d[pos]};
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            unsigned int row = pos / width_d;
            unsigned int col = pos % width_d;
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = {static_cast<uint16_t>(row), static_cast<uint16_t>(col), albedo_d[pos], ndvi_d[pos], surface_temperature_d[pos]};
        }
    }
}

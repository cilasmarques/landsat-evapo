#include "hip/hip_runtime.h"
#include "kernels.cuh"

__device__ int hotEndmemberLine_d;
__device__ int hotEndmemberCol_d;
__device__ int coldEndmemberLine_d;
__device__ int coldEndmemberCol_d;

__global__ void filter_valid_values(const float *target, float *filtered, int *ipos)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < height_d * width_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float value = target[pos];
        if (!isnan(value) && !isinf(value)) {
            int position = atomicAdd(ipos, 1);
            filtered[position] = value;
        }
    }
}

__global__ void process_pixels_STEEP(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surface_temperature_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviQuartileLow, float ndviQuartileHigh, float tsQuartileLow, float tsQuartileMid, float tsQuartileHigh, float albedoQuartileLow, float albedoQuartileMid, float albedoQuartileHigh)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviQuartileLow;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileMid && albedo_d[pos] < albedoQuartileHigh;
        bool hotTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] > tsQuartileMid && surface_temperature_d[pos] < tsQuartileHigh;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviQuartileHigh;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileLow && albedo_d[pos] < albedoQuartileMid;
        bool coldTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] < tsQuartileLow;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surface_temperature_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surface_temperature_d[pos], row, col);
        }
    }
}

__global__ void process_pixels_ASEBAL(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surface_temperature_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviHOTQuartile, float ndviCOLDQuartile, float tsHOTQuartile, float tsCOLDQuartile, float albedoHOTQuartile, float albedoCOLDQuartile)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviHOTQuartile;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoHOTQuartile;
        bool hotTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] > tsHOTQuartile;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviCOLDQuartile;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] < albedoCOLDQuartile;
        bool coldTS = !isnan(surface_temperature_d[pos]) && surface_temperature_d[pos] < tsCOLDQuartile;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surface_temperature_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surface_temperature_d[pos], row, col);
        }
    }
}

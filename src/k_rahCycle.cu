#include "hip/hip_runtime.h"
#include "kernels.cuh"

__device__ float a_d;
__device__ float b_d;

__global__ void d0_kernel(float *pai_d, float *d0_d, float CD1, float HGHT)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float cd1_pai_root = sqrt(CD1 * pai_d[pos]);

        d0_d[pos] = HGHT * ((1 - (1 / cd1_pai_root)) + (pow(exp(1.0), -cd1_pai_root) / cd1_pai_root));
    }
}

__global__ void ustar_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float u10)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ustar_d[pos] = (u10 * VON_KARMAN) / logf((10 - d0_d[pos]) / zom_d[pos]);
    }
}

__global__ void ustar_kernel_ASEBAL(float *zom_d, float *ustar_d, float u200)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ustar_d[pos] = (u200 * VON_KARMAN) / logf(200 / zom_d[pos]);
    }
}

__global__ void zom_kernel_STEEP(float *d0_d, float *pai_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;
    float CD = 0.01;
    float CR = 0.35;
    float PSICORR = 0.2;

    if (pos < width_d * height_d) {
        float gama = pow((CD + CR * (pai_d[pos] / 2)), -0.5);
        if (gama < 3.3)
            gama = 3.3;

        zom_d[pos] = (HGHT - d0_d[pos]) * exp(-VON_KARMAN * gama) + PSICORR;
    }
}

__global__ void zom_kernel_ASEBAL(float *ndvi_d, float *albedo_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        zom_d[pos] = exp((A_ZOM * ndvi_d[pos] / albedo_d[pos]) + B_ZOM);
    }
}

__global__ void kb_kernel(float *zom_d, float *ustar_d, float *pai_d, float *kb1_d, float *ndvi_d, float ndvi_max, float ndvi_min)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;

    float VON_KARMAN = 0.41;
    float visc = 0.00001461;
    float pr = 0.71;
    float c1 = 0.320;
    float c2 = 0.264;
    float c3 = 15.1;
    float cd = 0.2;
    float ct = 0.01;
    float sf_c = 0.3;
    float sf_d = 2.5;
    float sf_e = 4.0;
    float soil_moisture_day_rel = 0.33;

    if (pos < width_d * height_d) {
        float fc = 1 - pow((ndvi_d[pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fs = 1 - fc;

        float Re = (ustar_d[pos] * 0.009) / visc;
        float Ct = pow(pr, -0.667) * pow(Re, -0.5);
        float ratio = c1 - c2 * (exp(cd * -c3 * pai_d[pos]));
        float nec = (cd * pai_d[pos]) / (ratio * ratio * 2);
        float kbs = 2.46 * pow(Re, 0.25) - 2;

        float kb1_fst_part = (cd * VON_KARMAN) / (4 * ct * ratio * (1 - exp(nec * -0.5)));
        float kb1_sec_part = pow(fc, 2) + (VON_KARMAN * ratio * (zom_d[pos] / HGHT) / Ct);
        float kb1_trd_part = pow(fc, 2) * pow(fs, 2) + kbs * pow(fs, 2);
        float kb_ini = kb1_fst_part * kb1_sec_part * kb1_trd_part;

        float SF = sf_c + (1 / (1 + exp(sf_d - sf_e * soil_moisture_day_rel)));

        kb1_d[pos] = kb_ini * SF;
    }
}

__global__ void aerodynamic_resistance_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float *kb1_d, float *rah_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float rah_fst_part = 1 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = logf((10 - d0_d[pos]) / zom_d[pos]);
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        rah_d[pos] = (rah_fst_part * rah_sec_part) + rah_trd_part;
    }
}

__global__ void aerodynamic_resistance_kernel_ASEBAL(float *ustar_d, float *rah_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        rah_d[pos] = logf(2.0 / 0.1) / (ustar_d[pos] * VON_KARMAN);
    }
}

__global__ void rah_correction_cycle_STEEP(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surface_temperature_d, float *d0_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float ndvi_max, float ndvi_min)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float rah_ini_hot = rah_d[hot_pos];
        float rah_ini_cold = rah_d[cold_pos];

        float fc_hot = 1 - pow((ndvi_d[hot_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fc_cold = 1 - pow((ndvi_d[cold_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);

        float LE_hot = 0.55 * fc_hot * (net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos]) * 0.78;
        float LE_cold = 1.75 * fc_cold * (net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos]) * 0.78;

        float H_cold = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos] - LE_hot;
        float dt_cold = H_cold * rah_ini_cold / (RHO * SPECIFIC_HEAT_AIR);

        float H_hot = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos] - LE_cold;
        float dt_hot = H_hot * rah_ini_hot / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_hot - dt_cold) / (surface_temperature_d[hot_pos] - surface_temperature_d[cold_pos]);
        float a = dt_cold - (b * surface_temperature_d[cold_pos]);

        b_d = b;
        a_d = a;

        float dt_final = a + b * surface_temperature_d[pos];
        H_d[pos] = RHO * SPECIFIC_HEAT_AIR * dt_final / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surface_temperature_d[pos]) / (VON_KARMAN * GRAVITY * H_d[pos]));

        float y2 = pow((1 - (16 * (10 - d0_d[pos])) / L), 0.25);
        float x200 = pow((1 - (16 * (10 - d0_d[pos])) / L), 0.25);

        float psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi2 = -5 * ((10 - d0_d[pos]) / L);
            psi200 = -5 * ((10 - d0_d[pos]) / L);
        } else {
            psi2 = 2 * logf((1 + y2 * y2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        ustar_d[pos] = (VON_KARMAN * ustar_d[pos]) / (logf((10 - d0_d[pos]) / zom_d[pos]) - psi200);

        float rah_fst_part = 1 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = logf((10 - d0_d[pos]) / zom_d[pos]) - psi2;
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        rah_d[pos] = (rah_fst_part * rah_sec_part) + rah_trd_part;
    }
}

__global__ void rah_correction_cycle_ASEBAL(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surface_temperature_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float u200, int *stop_condition)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float rah_ini_hot = rah_d[hot_pos];
        float rah_ini_cold = rah_d[cold_pos];

        float H_cold = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos];
        float dt_cold = H_cold * rah_ini_cold / (RHO * SPECIFIC_HEAT_AIR);

        float H_hot = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos];
        float dt_hot = H_hot * rah_ini_hot / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_hot - dt_cold) / (surface_temperature_d[hot_pos] - surface_temperature_d[cold_pos]);
        float a = dt_cold - (b * surface_temperature_d[cold_pos]);

        b_d = b;
        a_d = a;

        float dt_final = a + b * (surface_temperature_d[pos]);

        H_d[pos] = RHO * SPECIFIC_HEAT_AIR * (dt_final) / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surface_temperature_d[pos]) / (VON_KARMAN * GRAVITY * H_d[pos]));

        float x1 = pow((1 - (16 * 0.1) / L), 0.25);
        float x2 = pow((1 - (16 * 2) / L), 0.25);
        float x200 = pow((1 - (16 * 200) / L), 0.25);

        float psi1, psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi1 = -5 * (0.1 / L);
            psi2 = -5 * (2 / L);
            psi200 = -5 * (2 / L);
        } else {
            psi1 = 2 * logf((1 + x1 * x1) / 2);
            psi2 = 2 * logf((1 + x2 * x2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        ustar_d[pos] = (VON_KARMAN * u200) / (logf(200 / zom_d[pos]) - psi200);
        rah_d[pos] = (logf(2 / 0.1) - psi2 + psi1) / (ustar_d[pos] * VON_KARMAN);

        if ((pos == hot_pos) && (fabsf(1 - (rah_ini_hot / rah_d[hot_pos])) < 0.05)) {
            atomicExch(stop_condition, 1);
        }
    }
}

__global__ void sensible_heat_flux_kernel(float *surface_temperature_d, float *rah_d, float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        sensible_heat_flux_d[pos] = RHO * SPECIFIC_HEAT_AIR * (a_d + b_d * surface_temperature_d[pos]) / rah_d[pos];
        if (!isnan(sensible_heat_flux_d[pos]) && sensible_heat_flux_d[pos] > (net_radiation_d[pos] - soil_heat_d[pos]))
            sensible_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];
    }
}

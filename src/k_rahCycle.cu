#include "hip/hip_runtime.h"
#include "constants.h"
#include "kernels.cuh"

__device__ float a_d;
__device__ float b_d;

__global__ void d0_kernel(float *pai_d, float *d0_d, float CD1, float HGHT)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float cd1_pai_root = sqrt(CD1 * pai_d[pos]);

        d0_d[pos] = HGHT * ((1.0 - (1.0 / cd1_pai_root)) + (exp(-cd1_pai_root) / cd1_pai_root));
    }
}

__global__ void ustar_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float u10)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ustar_d[pos] = (u10 * VON_KARMAN) / log((10.0 - d0_d[pos]) / zom_d[pos]);
    }
}

__global__ void ustar_kernel_ASEBAL(float *zom_d, float *ustar_d, float u200)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        ustar_d[pos] = (u200 * VON_KARMAN) / log(200.0 / zom_d[pos]);
    }
}

__global__ void zom_kernel_STEEP(float *d0_d, float *pai_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;
    float CD = 0.01;
    float CR = 0.35;
    float PSICORR = 0.2;

    if (pos < width_d * height_d) {
        float gama = pow((CD + CR * (pai_d[pos] / 2.0)), -0.5);
        if (gama < 3.3)
            gama = 3.3;

        zom_d[pos] = (HGHT - d0_d[pos]) * exp(-VON_KARMAN * gama) + PSICORR;
    }
}

__global__ void zom_kernel_ASEBAL(float *ndvi_d, float *albedo_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        zom_d[pos] = exp((A_ZOM * ndvi_d[pos] / albedo_d[pos]) + B_ZOM);
    }
}

__global__ void kb_kernel(float *zom_d, float *ustar_d, float *pai_d, float *kb1_d, float *ndvi_d, float ndvi_max, float ndvi_min)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;

    float VON_KARMAN = 0.41;
    float visc = 0.00001461;
    float pr = 0.71;
    float c1 = 0.320;
    float c2 = 0.264;
    float c3 = 15.1;
    float cd = 0.2;
    float ct = 0.01;
    float sf_c = 0.3;
    float sf_d = 2.5;
    float sf_e = 4.0;
    float soil_moisture_day_rel = 0.33;

    if (pos < width_d * height_d) {
        float fc = 1 - pow((ndvi_d[pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fs = 1 - fc;

        float Re = (ustar_d[pos] * 0.009) / visc;
        float Ct = pow(pr, -0.667) * pow(Re, -0.5);
        float ratio = c1 - c2 * (exp(cd * -c3 * pai_d[pos]));
        float nec = (cd * pai_d[pos]) / (ratio * ratio * 2.0);
        float kbs = 2.46 * pow(Re, 0.25) - 2.0;

        float kb1_fst_part = (cd * VON_KARMAN) / (4.0 * ct * ratio * (1.0 - exp(nec * -0.5)));
        float kb1_sec_part = pow(fc, 2) + (VON_KARMAN * ratio * (zom_d[pos] / HGHT) / Ct);
        float kb1_trd_part = pow(fc, 2) * pow(fs, 2) + kbs * pow(fs, 2);
        float kb_ini = kb1_fst_part * kb1_sec_part * kb1_trd_part;

	float SF = sf_c + (1.0 / (1.0 + exp(-sf_d) - sf_e * soil_moisture_day_rel));

        kb1_d[pos] = kb_ini * SF;
    }
}

__global__ void aerodynamic_resistance_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float *kb1_d, float *rah_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float rah_fst_part = 1.0 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = log((10.0 - d0_d[pos]) / zom_d[pos]);
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        rah_d[pos] = (rah_fst_part * rah_sec_part) + rah_trd_part;
    }
}

__global__ void aerodynamic_resistance_kernel_ASEBAL(float *ustar_d, float *rah_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        rah_d[pos] = log(2.0 / 0.1) / (ustar_d[pos] * VON_KARMAN);
    }
}

__global__ void rah_correction_cycle_STEEP(float *surface_temperature_d, float *d0_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float dt_final = a_d + b_d * surface_temperature_d[pos];
        H_d[pos] = RHO * SPECIFIC_HEAT_AIR * dt_final / rah_d[pos];
        float L = -1.0 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3.0) * surface_temperature_d[pos]) / (VON_KARMAN * GRAVITY * H_d[pos]));

        float y2 = pow((1.0 - (16.0 * (10.0 - d0_d[pos])) / L), 0.25);
        float x200 = pow((1.0 - (16.0 * (10.0 - d0_d[pos])) / L), 0.25);

        float psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi2 = -5.0 * ((10.0 - d0_d[pos]) / L);
            psi200 = -5.0 * ((10.0 - d0_d[pos]) / L);
        } else {
            psi2 = 2.0 * log((1.0 + y2 * y2) / 2.0);
            psi200 = 2.0 * log((1.0 + x200) / 2.0) + log((1.0 + x200 * x200) / 2.0) - 2.0 * atan(x200) + 0.5 * PI;
        }

        ustar_d[pos] = (VON_KARMAN * ustar_d[pos]) / (log((10.0 - d0_d[pos]) / zom_d[pos]) - psi200);

        float rah_fst_part = 1.0 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = log((10.0 - d0_d[pos]) / zom_d[pos]) - psi2;
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        rah_d[pos] = (rah_fst_part * rah_sec_part) + rah_trd_part;
    }
}

__global__ void rah_correction_cycle_ASEBAL(float *surface_temperature_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float u200)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        float dt_final = a_d + b_d * (surface_temperature_d[pos]);
        H_d[pos] = RHO * SPECIFIC_HEAT_AIR * (dt_final) / rah_d[pos];
        float L = -1.0 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3.0) * surface_temperature_d[pos]) / (VON_KARMAN * GRAVITY * H_d[pos]));

        float x1 = pow((1.0 - (16.0 * 0.1) / L), 0.25);
        float x2 = pow((1.0 - (16.0 * 2.0) / L), 0.25);
        float x200 = pow((1.0 - (16.0 * 200.0) / L), 0.25);

        float psi1, psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi1 = -5.0 * (0.1 / L);
            psi2 = -5.0 * (2.0 / L);
            psi200 = -5.0 * (2.0 / L);
        } else {
            psi1 = 2.0 * log((1.0 + x1 * x1) / 2.0);
            psi2 = 2.0 * log((1.0 + x2 * x2) / 2.0);
            psi200 = 2.0 * log((1.0 + x200) / 2.0) + log((1.0 + x200 * x200) / 2.0) - 2.0 * atan(x200) + 0.5 * PI;
        }

        ustar_d[pos] = (VON_KARMAN * u200) / (log(200.0 / zom_d[pos]) - psi200);
        rah_d[pos] = (log(2.0 / 0.1) - psi2 + psi1) / (ustar_d[pos] * VON_KARMAN);
    }
}

__global__ void sensible_heat_flux_kernel(float *surface_temperature_d, float *rah_d, float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d)
{
    unsigned int pos = threadIdx.x + blockIdx.x * blockDim.x;

    if (pos < width_d * height_d) {
        sensible_heat_flux_d[pos] = RHO * SPECIFIC_HEAT_AIR * (a_d + b_d * surface_temperature_d[pos]) / rah_d[pos];
        if (!isnan(sensible_heat_flux_d[pos]) && sensible_heat_flux_d[pos] > (net_radiation_d[pos] - soil_heat_d[pos]))
            sensible_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];
    }
}

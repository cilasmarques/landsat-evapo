#include "hip/hip_runtime.h"
#include "kernels.cuh"

__shared__ float a_d;
__shared__ float b_d;

__device__ int width_d;
__device__ int height_d;

__device__ int hotEndmemberLine_d;
__device__ int hotEndmemberCol_d;
__device__ int coldEndmemberLine_d;
__device__ int coldEndmemberCol_d;

__global__ void rad_kernel(float *band_d, float *radiance_d, float *rad_add_d, float *rad_mult_d, int band_idx)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        radiance_d[pos] = band_d[pos] * rad_mult_d[band_idx] + rad_add_d[band_idx];

        if (radiance_d[pos] <= 0)
            radiance_d[pos] = NAN;
    }
}

__global__ void ref_kernel(float *band_d, float *reflectance_d, float *ref_add_d, float *ref_mult_d, float sin_sun, int band_idx)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        reflectance_d[pos] = (band_d[pos] * ref_mult_d[band_idx] + ref_add_d[band_idx]) / sin_sun;

        if (reflectance_d[pos] <= 0)
            reflectance_d[pos] = NAN;
    }
}

__global__ void albedo_kernel(float *reflectance_blue_d, float *reflectance_green_d, float *reflectance_red_d, float *reflectance_nir_d, float *reflectance_swir1_d, float *reflectance_swir2_d, float *tal_d, float *albedo_d, float *ref_w_coeff_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float alb_toa = reflectance_blue_d[pos] * ref_w_coeff_d[PARAM_BAND_BLUE_INDEX] +
                        reflectance_green_d[pos] * ref_w_coeff_d[PARAM_BAND_GREEN_INDEX] +
                        reflectance_red_d[pos] * ref_w_coeff_d[PARAM_BAND_RED_INDEX] +
                        reflectance_nir_d[pos] * ref_w_coeff_d[PARAM_BAND_NIR_INDEX] +
                        reflectance_swir1_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR1_INDEX] +
                        reflectance_swir2_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR2_INDEX];

        albedo_d[pos] = (alb_toa - 0.03) / (tal_d[pos] * tal_d[pos]);

        if (albedo_d[pos] <= 0)
            albedo_d[pos] = NAN;
    }
}

__global__ void ndvi_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *ndvi_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ndvi_d[pos] = (reflectance_nir_d[pos] - reflectance_red_d[pos]) / (reflectance_nir_d[pos] + reflectance_red_d[pos]);

        if (ndvi_d[pos] <= -1 || ndvi_d[pos] >= 1)
            ndvi_d[pos] = NAN;
    }
}

__global__ void pai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *pai_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        pai_d[pos] = 10.1 * (reflectance_nir_d[pos] - sqrt(reflectance_red_d[pos])) + 3.1;

        if (pai_d[pos] < 0)
            pai_d[pos] = 0;
    }
}

__global__ void lai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *lai_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float savi = ((1 + 0.5) * (reflectance_nir_d[pos] - reflectance_red_d[pos])) / (0.5 + (reflectance_nir_d[pos] + reflectance_red_d[pos]));

        if (!isnan(savi) && savi > 0.687)
            lai_d[pos] = 6;
        if (!isnan(savi) && savi <= 0.687)
            lai_d[pos] = -logf((0.69 - savi) / 0.59) / 0.91;
        if (!isnan(savi) && savi < 0.1)
            lai_d[pos] = 0;

        if (lai_d[pos] < 0)
            lai_d[pos] = 0;
    }
}

__global__ void enb_kernel(float *lai_d, float *ndvi_d, float *enb_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] > 0)
            enb_d[pos] = (lai_d[pos] < 3) ? 0.97 + 0.0033 * lai_d[pos] : 0.98;            
        else if (ndvi_d[pos] < 0)
            enb_d[pos] = 0.99;
        else
            enb_d[pos] = NAN;
    }
}

__global__ void eo_kernel(float *lai_d, float *ndvi_d, float *eo_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] > 0)
            eo_d[pos] = (lai_d[pos] < 3) ? 0.95 + 0.01 * lai_d[pos] : 0.98;            
        else if (ndvi_d[pos] < 0)
            eo_d[pos] = 0.985;
        else
            eo_d[pos] = NAN;
    }
}

__global__ void ea_kernel(float *tal_d, float *ea_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        ea_d[pos] = 0.85 * pow((-1 * logf(tal_d[pos])), 0.09);
    }
}

__global__ void surface_temperature_kernel(float *enb_d, float *radiance_termal_d, float *surface_temperature_d, float k1, float k2)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        surface_temperature_d[pos] = k2 / (logf((enb_d[pos] * k1 / radiance_termal_d[pos]) + 1));

        if (surface_temperature_d[pos] < 0)
            surface_temperature_d[pos] = 0;
    }
}

__global__ void short_wave_radiation_kernel(float *tal_d, float *short_wave_radiation_d, float sun_elevation, float distance_earth_sun, float pi)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        short_wave_radiation_d[pos] = (1367 * sin(sun_elevation * pi / 180) * tal_d[pos]) / (distance_earth_sun * distance_earth_sun);
    }
}

__global__ void large_wave_radiation_surface_kernel(float *surface_temperature_d, float *eo_d, float *large_wave_radiation_surface_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        float temperature_pixel = surface_temperature_d[pos];
        float surface_temperature_pow_4 = temperature_pixel * temperature_pixel * temperature_pixel * temperature_pixel;
        large_wave_radiation_surface_d[pos] = eo_d[pos] * 5.67 * 1e-8 * surface_temperature_pow_4;
    }
}

__global__ void large_wave_radiation_atmosphere_kernel(float *ea_d, float *large_wave_radiation_atmosphere_d, float temperature)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float temperature_kelvin_pow_4 = temperature * temperature * temperature * temperature;
    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        large_wave_radiation_atmosphere_d[pos] = ea_d[pos] * 5.67 * 1e-8 * temperature_kelvin_pow_4;
    }
}

__global__ void net_radiation_kernel(float *short_wave_radiation_d, float *albedo_d, float *large_wave_radiation_atmosphere_d, float *large_wave_radiation_surface_d, float *eo_d, float *net_radiation_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        net_radiation_d[pos] = (1 - albedo_d[pos]) * short_wave_radiation_d[pos] + large_wave_radiation_atmosphere_d[pos] - large_wave_radiation_surface_d[pos] - (1 - eo_d[pos]) * large_wave_radiation_atmosphere_d[pos];

        if (net_radiation_d[pos] < 0)
            net_radiation_d[pos] = 0;
    }
}

__global__ void soil_heat_kernel(float *ndvi_d, float *albedo_d, float *surface_temperature_d, float *net_radiation_d, float *soil_heat_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] >= 0) {
            float temperature_celcius = surface_temperature_d[pos] - 273.15;
            float ndvi_pixel_pow_4 = ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos];
            soil_heat_d[pos] = temperature_celcius * (0.0038 + 0.0074 * albedo_d[pos]) * (1 - 0.98 * ndvi_pixel_pow_4) * net_radiation_d[pos];
        } else
            soil_heat_d[pos] = 0.5 * net_radiation_d[pos];

        if (soil_heat_d[pos] < 0)
            soil_heat_d[pos] = 0;
    }
}

__global__ void d0_kernel(float *pai_d, float *d0_d, float CD1, float HGHT)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        float cd1_pai_root = sqrt(CD1 * pai_d[pos]);

        d0_d[pos] = HGHT * ((1 - (1 / cd1_pai_root)) + (pow(exp(1.0), -cd1_pai_root) / cd1_pai_root));
    }
}

__global__ void ustar_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float u10)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        ustar_d[pos] = (u10 * VON_KARMAN) / logf((10 - d0_d[pos]) / zom_d[pos]);
    }
}

__global__ void ustar_kernel_ASEBAL(float *zom_d, float *ustar_d, float u200)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        ustar_d[pos] = (u200 * VON_KARMAN) / logf(200 / zom_d[pos]);
    }
}

__global__ void zom_kernel_STEEP(float *d0_d, float *pai_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;
    float CD = 0.01;
    float CR = 0.35;
    float PSICORR = 0.2;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float gama = pow((CD + CR * (pai_d[pos] / 2)), -0.5);
        if (gama < 3.3)
            gama = 3.3;

        zom_d[pos] = (HGHT - d0_d[pos]) * exp(-VON_KARMAN * gama) + PSICORR;
    }
}

__global__ void zom_kernel_ASEBAL(float *ndvi_d, float *albedo_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        zom_d[pos] = exp((A_ZOM * ndvi_d[pos] / albedo_d[pos]) + B_ZOM);
    }
}

__global__ void kb_kernel(float *zom_d, float *ustar_d, float *pai_d, float *kb1_d, float *ndvi_d, float ndvi_max, float ndvi_min)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float HGHT = 4;

    float VON_KARMAN = 0.41;
    float visc = 0.00001461;
    float pr = 0.71;
    float c1 = 0.320;
    float c2 = 0.264;
    float c3 = 15.1;
    float cd = 0.2;
    float ct = 0.01;
    float sf_c = 0.3;
    float sf_d = 2.5;
    float sf_e = 4.0;
    float soil_moisture_day_rel = 0.33;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float fc = 1 - pow((ndvi_d[pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fs = 1 - fc;

        float Re = (ustar_d[pos] * 0.009) / visc;
        float Ct = pow(pr, -(2/3)) * pow(Re, -(1/2));
        float ratio = c1 - c2 * (exp(cd * -c3 * pai_d[pos]));
        float nec = (cd * pai_d[pos]) / (ratio * ratio * 2);
        float kbs = 2.46 * pow(Re, 0.25) - 2;
    
        float kb1_fst_part = (cd * VON_KARMAN) / (4 * ct * ratio * (1 - exp(nec * -0.5)));
        float kb1_sec_part = pow(fc, 2) + (VON_KARMAN * ratio * (zom_d[pos] / HGHT) / Ct);
        float kb1_trd_part = pow(fc, 2) * pow(fs, 2) + kbs * pow(fs, 2);
        float kb_ini = kb1_fst_part * kb1_sec_part * kb1_trd_part;

        float SF = sf_c + (1 / (1 + exp(sf_d - sf_e * soil_moisture_day_rel)));

        kb1_d[pos] = kb_ini * SF;
    }
}

__global__ void aerodynamic_resistance_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float *kb1_d, float *rah_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float rah_fst_part = 1 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = logf((10 - d0_d[pos]) / zom_d[pos]);
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        rah_d[pos] = (rah_fst_part * rah_sec_part) + rah_trd_part;
    }
}

__global__ void aerodynamic_resistance_kernel_ASEBAL(float *ustar_d, float *rah_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        rah_d[pos] = logf(2.0 / 0.1) / (ustar_d[pos] * VON_KARMAN);
    }
}

__global__ void sensible_heat_flux_kernel(float *surface_temperature_d, float *rah_d, float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        sensible_heat_flux_d[pos] = RHO * SPECIFIC_HEAT_AIR * (a_d + b_d * (surface_temperature_d[pos])) / rah_d[pos];
        if (!isnan(sensible_heat_flux_d[pos]) && sensible_heat_flux_d[pos] > (net_radiation_d[pos] - soil_heat_d[pos]))
            sensible_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];
    }
}

__global__ void latent_heat_flux_kernel(float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d, float *latent_heat_flux_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        latent_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos] - sensible_heat_flux_d[pos];
    }
}

__global__ void net_radiation_24h_kernel(float *albedo_d, float Rs24h, float Ra24h, float *net_radiation_24h_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    int FL = 110;
    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        net_radiation_24h_d[pos] = (1 - albedo_d[pos]) * Rs24h - FL * Rs24h / Ra24h;
    }
}

__global__ void evapotranspiration_24h_kernel(float *surface_temperature_d, float *latent_heat_flux_d, float *net_radiation_d, float *soil_heat_d, float *net_radiation_24h_d, float *evapotranspiration_24h_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        float temperature_celcius = surface_temperature_d[pos] - 273.15;
        evapotranspiration_24h_d[pos] = (86400 / ((2.501 - 0.0236 * temperature_celcius) * pow(10, 6))) * (latent_heat_flux_d[pos] / (net_radiation_d[pos] - soil_heat_d[pos])) * net_radiation_24h_d[pos];
    }
}

__global__ void rah_correction_cycle_STEEP(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surf_temp_d, float *d0_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float ndvi_max, float ndvi_min)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float rah_ini_hot = rah_d[hot_pos];
        float rah_ini_cold = rah_d[cold_pos];

        float fc_hot = 1 - pow((ndvi_d[hot_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fc_cold = 1 - pow((ndvi_d[cold_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);

        float LE_hot = 0.55 * fc_hot * (net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos]) * 0.78;
        float LE_cold = 1.75 * fc_cold * (net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos]) * 0.78;

        float H_cold = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos] - LE_hot;
        float dt_cold = H_cold * rah_ini_cold / (RHO * SPECIFIC_HEAT_AIR);

        float H_hot = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos] - LE_cold;
        float dt_hot = H_hot * rah_ini_hot / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_hot - dt_cold) / (surf_temp_d[hot_pos] - surf_temp_d[cold_pos]);
        float a = dt_cold - (b * surf_temp_d[cold_pos]);

        b_d = b;
        a_d = a;

        float dt_final = a + (b * surf_temp_d[pos]);

        float sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dt_final) / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surf_temp_d[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

        float DISP = d0_d[pos];
        float y2 = pow((1 - (16 * (10 - d0_d[pos])) / L), 0.25);
        float x200 = pow((1 - (16 * (10 - DISP)) / L), 0.25);

        float psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi2 = -5 * ((10 - DISP) / L);
            psi200 = -5 * ((10 - DISP) / L);
        } else {
            psi2 = 2 * logf((1 + y2 * y2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        float ust = (VON_KARMAN * ustar_d[pos]) / (logf((10 - DISP) / zom_d[pos]) - psi200);

        float rah_fst_part = 1 / (ustar_d[pos] * VON_KARMAN);
        float rah_sec_part = logf((10 - d0_d[pos]) / zom_d[pos]) - psi2;
        float rah_trd_part = rah_fst_part * kb1_d[pos];
        float rah = (rah_fst_part * rah_sec_part) + rah_trd_part;

        ustar_d[pos] = ust;
        rah_d[pos] = rah;
        H_d[pos] = sensibleHeatFlux;
    }
}

__global__ void rah_correction_cycle_ASEBAL(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surf_temp_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float ndvi_max, float ndvi_min, float u200, int *stop_condition)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float rah_ini_hot = rah_d[hot_pos];
        float rah_ini_cold = rah_d[cold_pos];

        float H_cold = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos];
        float dt_cold = H_cold * rah_ini_cold / (RHO * SPECIFIC_HEAT_AIR);

        float H_hot = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos];
        float dt_hot = H_hot * rah_ini_hot / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_hot - dt_cold) / (surf_temp_d[hot_pos] - surf_temp_d[cold_pos]);
        float a = dt_cold - (b * surf_temp_d[cold_pos]);

        b_d = b;
        a_d = a;

        float dt_final = a + b * (surf_temp_d[pos]);

        float sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dt_final) / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surf_temp_d[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

        float x1 = pow((1 - (16 * 0.1) / L), 0.25);
        float x2 = pow((1 - (16 * 2) / L), 0.25);
        float x200 = pow((1 - (16 * 200) / L), 0.25);

        float psi1, psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi1 = -5 * (0.1 / L);
            psi2 = -5 * (2 / L);
            psi200 = -5 * (2 / L);
        } else {
            psi1 = 2 * logf((1 + x1 * x1) / 2);
            psi2 = 2 * logf((1 + x2 * x2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        float ust = (VON_KARMAN * u200) / (logf(200 / zom_d[pos]) - psi200);
        float rah = (logf(2 / 0.1) - psi2 + psi1) / (ustar_d[pos] * VON_KARMAN);

        if ((pos == hot_pos) && (fabsf(1 - (rah_ini_hot / rah)) < 0.05)) {
            atomicExch(stop_condition, 1);
        }

        ustar_d[pos] = ust;
        rah_d[pos] = rah;
        H_d[pos] = sensibleHeatFlux;
    }
}

__global__ void filter_valid_values(const float *target, float *filtered, int *ipos)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < height_d * width_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float value = target[pos];
        if (!isnan(value) && !isinf(value)) {
            int position = atomicAdd(ipos, 1);
            filtered[position] = value;
        }
    }
}

__global__ void process_pixels_STEEP(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surf_temp_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviQuartileLow, float ndviQuartileHigh, float tsQuartileLow, float tsQuartileMid, float tsQuartileHigh, float albedoQuartileLow, float albedoQuartileMid, float albedoQuartileHigh)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviQuartileLow;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileMid && albedo_d[pos] < albedoQuartileHigh;
        bool hotTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] > tsQuartileMid && surf_temp_d[pos] < tsQuartileHigh;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviQuartileHigh;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileLow && albedo_d[pos] < albedoQuartileMid;
        bool coldTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] < tsQuartileLow;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }
    }
}

__global__ void process_pixels_ASEBAL(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surf_temp_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviHOTQuartile, float ndviCOLDQuartile, float tsHOTQuartile, float tsCOLDQuartile, float albedoHOTQuartile, float albedoCOLDQuartile)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviHOTQuartile;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoHOTQuartile;
        bool hotTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] > tsHOTQuartile;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviCOLDQuartile;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] < albedoCOLDQuartile;
        bool coldTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] < tsCOLDQuartile;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }
    }
}

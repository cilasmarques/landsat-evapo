#include "hip/hip_runtime.h"
#include "kernels.cuh"

__shared__ float a_d;
__shared__ float b_d;

__device__ int width_d;
__device__ int height_d;

__device__ int hotEndmemberLine_d;
__device__ int hotEndmemberCol_d;
__device__ int coldEndmemberLine_d;
__device__ int coldEndmemberCol_d;

__global__ void rad_kernel(float *band_d, float *radiance_d, float *rad_add_d, float *rad_mult_d, int band_idx)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        radiance_d[pos] = band_d[pos] * rad_mult_d[band_idx] + rad_add_d[band_idx];

        if (radiance_d[pos] <= 0)
            radiance_d[pos] = NAN;
    }
}

__global__ void ref_kernel(float *band_d, float *reflectance_d, float *ref_add_d, float *ref_mult_d, float sin_sun, int band_idx)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        reflectance_d[pos] = (band_d[pos] * ref_mult_d[band_idx] + ref_add_d[band_idx]) / sin_sun;

        if (reflectance_d[pos] <= 0)
            reflectance_d[pos] = NAN;
    }
}

__global__ void albedo_kernel(float *reflectance_blue_d, float *reflectance_green_d, float *reflectance_red_d, float *reflectance_nir_d, float *reflectance_swir1_d, float *reflectance_swir2_d, float *tal_d, float *albedo_d, float *ref_w_coeff_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        float alb_toa = reflectance_blue_d[pos] * ref_w_coeff_d[PARAM_BAND_BLUE_INDEX] +
                        reflectance_green_d[pos] * ref_w_coeff_d[PARAM_BAND_GREEN_INDEX] +
                        reflectance_red_d[pos] * ref_w_coeff_d[PARAM_BAND_RED_INDEX] +
                        reflectance_nir_d[pos] * ref_w_coeff_d[PARAM_BAND_NIR_INDEX] +
                        reflectance_swir1_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR1_INDEX] +
                        reflectance_swir2_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR2_INDEX];

        albedo_d[pos] = (alb_toa - 0.03) / (tal_d[pos] * tal_d[pos]);

        if (albedo_d[pos] <= 0)
            albedo_d[pos] = NAN;
    }
}

__global__ void ndvi_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *ndvi_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        ndvi_d[pos] = (reflectance_nir_d[pos] - reflectance_red_d[pos]) / (reflectance_nir_d[pos] + reflectance_red_d[pos]);

        if (ndvi_d[pos] <= -1 || ndvi_d[pos] >= 1)
            ndvi_d[pos] = NAN;
    }
}

__global__ void pai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *pai_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        pai_d[pos] = 10.1 * (reflectance_nir_d[pos] - sqrt(reflectance_red_d[pos])) + 3.1;

        if (pai_d[pos] < 0)
            pai_d[pos] = 0;
    }
}

__global__ void lai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *lai_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        float savi = ((1 + 0.5) * (reflectance_nir_d[pos] - reflectance_red_d[pos])) / (0.5 + (reflectance_nir_d[pos] + reflectance_red_d[pos]));

        if (!isnan(savi) && savi > 0.687)
            lai_d[pos] = 6;
        if (!isnan(savi) && savi <= 0.687)
            lai_d[pos] = -logf((0.69 - savi) / 0.59) / 0.91;
        if (!isnan(savi) && savi < 0.1)
            lai_d[pos] = 0;

        if (lai_d[pos] < 0)
            lai_d[pos] = 0;
    }
}

__global__ void enb_kernel(float *lai_d, float *ndvi_d, float *enb_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] > 0)
            enb_d[pos] = (lai_d[pos] < 3) ? 0.97 + 0.0033 * lai_d[pos] : 0.98;            
        else if (ndvi_d[pos] < 0)
            enb_d[pos] = 0.99;
        else
            enb_d[pos] = NAN;
    }
}

__global__ void eo_kernel(float *lai_d, float *ndvi_d, float *eo_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] > 0)
            eo_d[pos] = (lai_d[pos] < 3) ? 0.95 + 0.01 * lai_d[pos] : 0.98;            
        else if (ndvi_d[pos] < 0)
            eo_d[pos] = 0.985;
        else
            eo_d[pos] = NAN;
    }
}

__global__ void ea_kernel(float *tal_d, float *ea_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        ea_d[pos] = 0.85 * pow((-1 * logf(tal_d[pos])), 0.09);
    }
}

__global__ void surface_temperature_kernel(float *enb_d, float *radiance_termal_d, float *surface_temperature_d, float k1, float k2)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        surface_temperature_d[pos] = k2 / (logf((enb_d[pos] * k1 / radiance_termal_d[pos]) + 1));

        if (surface_temperature_d[pos] < 0)
            surface_temperature_d[pos] = 0;
    }
}

__global__ void short_wave_radiation_kernel(float *tal_d, float *short_wave_radiation_d, float sun_elevation, float distance_earth_sun, float pi)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        short_wave_radiation_d[pos] = (1367 * sin(sun_elevation * pi / 180) * tal_d[pos]) / (distance_earth_sun * distance_earth_sun);
    }
}

__global__ void large_wave_radiation_surface_kernel(float *surface_temperature_d, float *eo_d, float *large_wave_radiation_surface_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        float temperature_pixel = surface_temperature_d[pos];
        float surface_temperature_pow_4 = temperature_pixel * temperature_pixel * temperature_pixel * temperature_pixel;
        large_wave_radiation_surface_d[pos] = eo_d[pos] * 5.67 * 1e-8 * surface_temperature_pow_4;
    }
}

__global__ void large_wave_radiation_atmosphere_kernel(float *ea_d, float *large_wave_radiation_atmosphere_d, float temperature)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    float temperature_kelvin_pow_4 = temperature * temperature * temperature * temperature;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        large_wave_radiation_atmosphere_d[pos] = ea_d[pos] * 5.67 * 1e-8 * temperature_kelvin_pow_4;
    }
}

__global__ void net_radiation_kernel(float *short_wave_radiation_d, float *albedo_d, float *large_wave_radiation_atmosphere_d, float *large_wave_radiation_surface_d, float *eo_d, float *net_radiation_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        net_radiation_d[pos] = (1 - albedo_d[pos]) * short_wave_radiation_d[pos] + large_wave_radiation_atmosphere_d[pos] - large_wave_radiation_surface_d[pos] - (1 - eo_d[pos]) * large_wave_radiation_atmosphere_d[pos];

        if (net_radiation_d[pos] < 0)
            net_radiation_d[pos] = 0;
    }
}

__global__ void soil_heat_kernel(float *ndvi_d, float *albedo_d, float *surface_temperature_d, float *net_radiation_d, float *soil_heat_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        if (ndvi_d[pos] >= 0) {
            float temperature_celcius = surface_temperature_d[pos] - 273.15;
            float ndvi_pixel_pow_4 = ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos];
            soil_heat_d[pos] = temperature_celcius * (0.0038 + 0.0074 * albedo_d[pos]) * (1 - 0.98 * ndvi_pixel_pow_4) * net_radiation_d[pos];
        } else
            soil_heat_d[pos] = 0.5 * net_radiation_d[pos];

        if (soil_heat_d[pos] < 0)
            soil_heat_d[pos] = 0;
    }
}

__global__ void d0_kernel(float *pai_d, float *d0_d, float CD1, float HGHT)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        float cd1_pai_root = sqrt(CD1 * pai_d[pos]);

        d0_d[pos] = HGHT * ((1 - (1 / cd1_pai_root)) + (pow(exp(1.0), -cd1_pai_root) / cd1_pai_root));
    }
}

__global__ void ustar_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float u10)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    float zu = 10;
    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        ustar_d[pos] = (u10 * VON_KARMAN) / logf((zu - d0_d[pos]) / zom_d[pos]);
    }
}

__global__ void ustar_kernel_ASEBAL(float *zom_d, float *ustar_d, float u200)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        ustar_d[pos] = (u200 * VON_KARMAN) / logf(200 / zom_d[pos]);
    }
}

__global__ void zom_kernel_STEEP(float *d0_d, float *pai_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    float HGHT = 4;
    float CD = 0.01;
    float CR = 0.35;
    float PSICORR = 0.2;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        float gama = pow((CD + CR * (pai_d[pos] / 2)), -0.5);

        if (gama < 3.3)
            gama = 3.3;

        zom_d[pos] = (HGHT - d0_d[pos]) * pow(exp(1.0), (-VON_KARMAN * gama) + PSICORR);
    }
}

__global__ void zom_kernel_ASEBAL(float *ndvi_d, float *zom_d, float A_ZOM, float B_ZOM)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        zom_d[pos] = exp(A_ZOM + B_ZOM * ndvi_d[pos]);
    }
}

__global__ void kb_kernel(float *zom_d, float *ustar_d, float *pai_d, float *kb1_d, float *ndvi_d, float ndvi_max, float ndvi_min)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    float HGHT = 4;

    float VON_KARMAN = 0.41;
    float visc = 0.00001461;
    float pr = 0.71;
    float c1 = 0.320;
    float c2 = 0.264;
    float c3 = 15.1;
    float cd = 0.2;
    float ct = 0.01;
    float sf_c = 0.3;
    float sf_d = 2.5;
    float sf_e = 4.0;
    float soil_moisture_day_rel = 0.33;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        float Re_star = (ustar_d[pos] * 0.009) / visc;
        float Ct_star = pow(pr, -0.667) * pow(Re_star, -0.5);
        float beta = c1 - c2 * (exp((cd * -c3 * pai_d[pos])));
        float nec_terra = (cd * pai_d[pos]) / (beta * beta * 2);

        float kb1_fst_part = (cd * VON_KARMAN) / (4 * ct * beta * (1 - exp(nec_terra * -0.5)));
        float kb1_sec_part = (beta * VON_KARMAN * (zom_d[pos] / HGHT)) / Ct_star;
        float kb1s = (pow(Re_star, 0.25) * 2.46) - 2;

        float fc = 1 - pow((ndvi_d[pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fs = 1 - fc;

        float SF = sf_c + (1 / (1 + pow(exp(1.0), (sf_d - (sf_e * soil_moisture_day_rel)))));

        kb1_d[pos] = ((kb1_fst_part * pow(fc, 2)) + (kb1_sec_part * pow(fc, 2) * pow(fs, 2)) + (pow(fs, 2) * kb1s)) * SF;
    }
}

__global__ void aerodynamic_resistance_kernel_STEEP(float *zom_d, float *d0_d, float *ustar_d, float *kb1_d, float *rah_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    float zu = 10.0;
    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float DISP = d0_d[pos];
        float zoh_terra = zom_d[pos] / pow(exp(1.0), (kb1_d[pos]));
        float temp_rah1_corr_terra = 1 / (ustar_d[pos] * VON_KARMAN);
        float temp_rah2_corr_terra = logf((zu - DISP) / zom_d[pos]);
        float temp_rah3_corr_terra = temp_rah1_corr_terra * logf(zom_d[pos] / zoh_terra);

        rah_d[pos] = (temp_rah1_corr_terra * temp_rah2_corr_terra) + temp_rah3_corr_terra;
    }
}

__global__ void aerodynamic_resistance_kernel_ASEBAL(float *ustar_d, float *rah_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;
        rah_d[pos] = logf(2.0 / 0.1) / (ustar_d[pos] * VON_KARMAN);
    }
}

__global__ void sensible_heat_flux_kernel(float *surface_temperature_d, float *rah_d, float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;

        sensible_heat_flux_d[pos] = RHO * SPECIFIC_HEAT_AIR * (a_d + b_d * (surface_temperature_d[pos])) / rah_d[pos];
        if (!isnan(sensible_heat_flux_d[pos]) && sensible_heat_flux_d[pos] > (net_radiation_d[pos] - soil_heat_d[pos])) {
            sensible_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];
        }
    }
}

__global__ void latent_heat_flux_kernel(float *net_radiation_d, float *soil_heat_d, float *sensible_heat_flux_d, float *latent_heat_flux_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        latent_heat_flux_d[pos] = net_radiation_d[pos] - soil_heat_d[pos] - sensible_heat_flux_d[pos];
    }
}

__global__ void net_radiation_24h_kernel(float *albedo_d, float Rs24h, float Ra24h, float *net_radiation_24h_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    int FL = 110;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        net_radiation_24h_d[pos] = (1 - albedo_d[pos]) * Rs24h - FL * Rs24h / Ra24h;
    }
}

__global__ void evapotranspiration_fraction_kernel(float *net_radiation_d, float *soil_heat_d, float *latent_heat_flux_d, float *evapotranspiration_fraction_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        evapotranspiration_fraction_d[pos] = latent_heat_flux_d[pos] / (net_radiation_d[pos] - soil_heat_d[pos]);
    }
}

__global__ void sensible_heat_flux_24h_kernel(float *net_radiation_24h_d, float *evapotranspiration_fraction_d, float *sensible_heat_flux_24h_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        sensible_heat_flux_24h_d[pos] = (1 - evapotranspiration_fraction_d[pos]) * net_radiation_24h_d[pos];
    }
}

__global__ void latent_heat_flux_24h_kernel(float *net_radiation_24h_d, float *evapotranspiration_fraction_d, float *latent_heat_flux_24h_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        latent_heat_flux_24h_d[pos] = evapotranspiration_fraction_d[pos] * net_radiation_24h_d[pos];
    }
}

__global__ void evapotranspiration_24h_kernel(float *latent_heat_flux_24h_d, float *evapotranspiration_24h_d, float v7_max, float v7_min)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        evapotranspiration_24h_d[pos] = (latent_heat_flux_24h_d[pos] * 86400) / ((2.501 - 0.00236 * (v7_max + v7_min) / 2) * 1e+6);
    }
}

__global__ void evapotranspiration_kernel(float *net_radiation_24h_d, float *evapotranspiration_fraction_d, float *evapotranspiration_d)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_d;
    unsigned int col = idx % width_d;

    if (idx < width_d * height_d) {
        unsigned int pos = row * width_d + col;
        evapotranspiration_d[pos] = net_radiation_24h_d[pos] * evapotranspiration_fraction_d[pos] * 0.035;
    }
}

__global__ void rah_correction_cycle_STEEP(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surf_temp_d, float *d0_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float ndvi_max, float ndvi_min)
{
    // Identify 1D position
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float fc_hot = 1 - pow((ndvi_d[hot_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
        float fc_cold = 1 - pow((ndvi_d[cold_pos] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);

        float rah_ini_pq_terra = rah_d[hot_pos];
        float rah_ini_pf_terra = rah_d[cold_pos];

        float LEc_terra = 0.55 * fc_hot * (net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos]) * 0.78;
        float LEc_terra_pf = 1.75 * fc_cold * (net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos]) * 0.78;

        float H_pf_terra = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos] - LEc_terra_pf;
        float dt_pf_terra = H_pf_terra * rah_ini_pf_terra / (RHO * SPECIFIC_HEAT_AIR);

        float H_pq_terra = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos] - LEc_terra;
        float dt_pq_terra = H_pq_terra * rah_ini_pq_terra / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_pq_terra - dt_pf_terra) / (surf_temp_d[hot_pos] - surf_temp_d[cold_pos]);
        float a = dt_pf_terra - (b * surf_temp_d[cold_pos]);

        b_d = b;
        a_d = a;

        float DISP = d0_d[pos];
        float dT_ini_terra = a + b * (surf_temp_d[pos]);

        float sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dT_ini_terra) / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surf_temp_d[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

        float y2 = pow((1 - (16 * (10 - DISP)) / L), 0.25);
        float x200 = pow((1 - (16 * (10 - DISP)) / L), 0.25);

        float psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi2 = -5 * ((10 - DISP) / L);
            psi200 = -5 * ((10 - DISP) / L);
        } else {
            psi2 = 2 * logf((1 + y2 * y2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        float ust = (VON_KARMAN * ustar_d[pos]) / (logf((10 - DISP) / zom_d[pos]) - psi200);

        float zoh_terra = zom_d[pos] / pow(exp(1.0), (kb1_d[pos] + psi200));
        float temp_rah1_corr_terra = 1 / (ustar_d[pos] * VON_KARMAN);
        float temp_rah2_corr_terra = logf((10 - DISP) / zom_d[pos]) - psi2;
        float temp_rah3_corr_terra = temp_rah1_corr_terra * logf(zom_d[pos] / zoh_terra);
        float rah = (temp_rah1_corr_terra * temp_rah2_corr_terra) + temp_rah3_corr_terra;

        ustar_d[pos] = ust;
        rah_d[pos] = rah;
        H_d[pos] = sensibleHeatFlux;
    }
}

__global__ void rah_correction_cycle_ASEBAL(float *net_radiation_d, float *soil_heat_flux_d, float *ndvi_d, float *surf_temp_d, float *kb1_d, float *zom_d, float *ustar_d, float *rah_d, float *H_d, float ndvi_max, float ndvi_min, float u200, int *stop_condition)
{
    // Identify 1D position
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        unsigned int hot_pos = hotEndmemberLine_d * width_d + hotEndmemberCol_d;
        unsigned int cold_pos = coldEndmemberLine_d * width_d + coldEndmemberCol_d;

        float rah_ini_pq_terra = rah_d[hot_pos];
        float rah_ini_pf_terra = rah_d[cold_pos];

        float H_pf_terra = net_radiation_d[cold_pos] - soil_heat_flux_d[cold_pos];
        float dt_pf_terra = H_pf_terra * rah_ini_pf_terra / (RHO * SPECIFIC_HEAT_AIR);

        float H_pq_terra = net_radiation_d[hot_pos] - soil_heat_flux_d[hot_pos];
        float dt_pq_terra = H_pq_terra * rah_ini_pq_terra / (RHO * SPECIFIC_HEAT_AIR);

        float b = (dt_pq_terra - dt_pf_terra) / (surf_temp_d[hot_pos] - surf_temp_d[cold_pos]);
        float a = dt_pf_terra - (b * (surf_temp_d[cold_pos]));

        b_d = b;
        a_d = a;

        float dT_ini_terra = a + b * (surf_temp_d[pos]);

        float sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dT_ini_terra) / rah_d[pos];
        float L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustar_d[pos], 3) * surf_temp_d[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

        float x1 = pow((1 - (16 * 0.1) / L), 0.25);
        float x2 = pow((1 - (16 * 2) / L), 0.25);
        float x200 = pow((1 - (16 * 200) / L), 0.25);

        float psi1, psi2, psi200;
        if (!isnan(L) && L > 0) {
            psi1 = -5 * (0.1 / L);
            psi2 = -5 * (2 / L);
            psi200 = -5 * (2 / L);
        } else {
            psi1 = 2 * logf((1 + x1 * x1) / 2);
            psi2 = 2 * logf((1 + x2 * x2) / 2);
            psi200 = 2 * logf((1 + x200) / 2) + logf((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
        }

        float ust = (VON_KARMAN * u200) / (logf(200 / zom_d[pos]) - psi200);
        float rah = (logf(2 / 0.1) - psi2 + psi1) / (ustar_d[pos] * VON_KARMAN);

        if ((pos == hot_pos) && (fabsf(1 - (rah_ini_pq_terra / rah)) < 0.05)) {
            atomicExch(stop_condition, 1);
        }

        ustar_d[pos] = ust;
        rah_d[pos] = rah;
        H_d[pos] = sensibleHeatFlux;
    }
}

__global__ void filter_valid_values(const float *target, float *filtered, int *ipos)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < height_d * width_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        float value = target[pos];
        if (!isnan(value) && !isinf(value)) {
            int position = atomicAdd(ipos, 1);
            filtered[position] = value;
        }
    }
}

__global__ void process_pixels_STEEP(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surf_temp_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviQuartileLow, float ndviQuartileHigh, float tsQuartileLow, float tsQuartileMid, float tsQuartileHigh, float albedoQuartileLow, float albedoQuartileMid, float albedoQuartileHigh)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviQuartileLow;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileMid && albedo_d[pos] < albedoQuartileHigh;
        bool hotTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] > tsQuartileMid && surf_temp_d[pos] < tsQuartileHigh;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviQuartileHigh;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoQuartileLow && albedo_d[pos] < albedoQuartileMid;
        bool coldTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] < tsQuartileLow;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }
    }
}

__global__ void process_pixels_ASEBAL(Endmember *hotCandidates_d, Endmember *coldCandidates_d, int *indexes_d, float *ndvi_d, float *surf_temp_d, float *albedo_d, float *net_radiation_d, float *soil_heat_d, float *ho_d, float ndviHOTQuartile, float ndviCOLDQuartile, float tsHOTQuartile, float tsCOLDQuartile, float albedoHOTQuartile, float albedoCOLDQuartile)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < width_d * height_d) {
        unsigned int row = idx / width_d;
        unsigned int col = idx % width_d;
        unsigned int pos = row * width_d + col;

        ho_d[pos] = net_radiation_d[pos] - soil_heat_d[pos];

        bool hotNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > 0.10 && ndvi_d[pos] < ndviHOTQuartile;
        bool hotAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] > albedoHOTQuartile;
        bool hotTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] > tsHOTQuartile;

        bool coldNDVI = !isnan(ndvi_d[pos]) && ndvi_d[pos] > ndviCOLDQuartile;
        bool coldAlbedo = !isnan(albedo_d[pos]) && albedo_d[pos] < albedoCOLDQuartile;
        bool coldTS = !isnan(surf_temp_d[pos]) && surf_temp_d[pos] < tsCOLDQuartile;

        if (hotAlbedo && hotNDVI && hotTS) {
            int ih = atomicAdd(&indexes_d[0], 1);
            hotCandidates_d[ih] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS) {
            int ic = atomicAdd(&indexes_d[1], 1);
            coldCandidates_d[ic] = Endmember(ndvi_d[pos], surf_temp_d[pos], row, col);
        }
    }
}

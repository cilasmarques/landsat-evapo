#include "hip/hip_runtime.h"
#include "endmembers.h"
#include "filter.cuh"

void compute_H0(float *net_radiation, float *soil_heat_flux, int height_band, int width_band, float *ho)
{
  for (int i = 0; i < height_band * width_band; i++)
    ho[i] = net_radiation[i] - soil_heat_flux[i];
};

void get_quartiles(float *target, float *v_quartile, int height_band, int width_band, float first_interval, float middle_interval, float last_interval)
{
  const int SIZE = height_band * width_band;
  float *target_values = (float *)malloc(sizeof(float) * SIZE);

  if (target_values == NULL)
    exit(15);

  int pos = 0;
  for (int i = 0; i < height_band * width_band; i++)
  {
    if (!isnan(target[i]) && !isinf(target[i]))
    {
      target_values[pos] = target[i];
      pos++;
    }
  }

  int first_index = static_cast<int>(floor(first_interval * pos));
  int middle_index = static_cast<int>(floor(middle_interval * pos));
  int last_index = static_cast<int>(floor(last_interval * pos));

  std::nth_element(target_values, target_values + first_index, target_values + pos);
  v_quartile[0] = target_values[first_index];

  std::nth_element(target_values, target_values + middle_index, target_values + pos);
  v_quartile[1] = target_values[middle_index];

  std::nth_element(target_values, target_values + last_index, target_values + pos);
  v_quartile[2] = target_values[last_index];

  free(target_values);
}

pair<Candidate, Candidate> getEndmembersSTEPP(float *ndvi, float *d_ndvi, float *surface_temperature, float *d_surface_temperature, float *albedo, float *d_albedo,
                                              float *net_radiation, float *d_net_radiation, float *soil_heat, float *d_soil_heat,
                                              int blocks_num, int threads_num, int height_band, int width_band)
{
  int hot_index, cold_index = 0;
  int *d_hot_index, *d_cold_index;
  hipMalloc((void **)&d_hot_index, sizeof(int));
  hipMalloc((void **)&d_cold_index, sizeof(int));
  hipMemcpy(d_hot_index, &hot_index, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_cold_index, &cold_index, sizeof(int), hipMemcpyHostToDevice);

  Candidate *hotCandidates, *coldCandidates;
  hotCandidates = (Candidate *)malloc(sizeof(Candidate) * height_band * width_band);
  coldCandidates = (Candidate *)malloc(sizeof(Candidate) * height_band * width_band);

  Candidate *d_hotCandidates, *d_coldCandidates;
  hipMalloc((void **)&d_hotCandidates, sizeof(Candidate) * height_band * width_band);
  hipMalloc((void **)&d_coldCandidates, sizeof(Candidate) * height_band * width_band);

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles(ndvi, ndviQuartile.data(), height_band, width_band, 0.15, 0.97, 0.97);
  get_quartiles(albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature, tsQuartile.data(), height_band, width_band, 0.20, 0.85, 0.97);

  float *ho = (float *)malloc(sizeof(float) * height_band * width_band);
  compute_H0(net_radiation, soil_heat, height_band, width_band, ho);

  float *d_ho;
  hipMalloc((void **)&d_ho, sizeof(float) * height_band * width_band);
  hipMemcpy(d_ho, ho, sizeof(float) * height_band * width_band, hipMemcpyHostToDevice);

  process_pixels<<<blocks_num, threads_num>>>(d_hotCandidates, d_coldCandidates, d_hot_index, d_cold_index,
                                              d_ndvi, d_surface_temperature, d_albedo, d_net_radiation, d_soil_heat, d_ho,
                                              ndviQuartile[0], ndviQuartile[1], tsQuartile[0], tsQuartile[1], tsQuartile[2],
                                              albedoQuartile[0], albedoQuartile[1], albedoQuartile[2], height_band, width_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  hipMemcpy(hotCandidates, d_hotCandidates, sizeof(Candidate) * height_band * width_band, hipMemcpyDeviceToHost);
  hipMemcpy(coldCandidates, d_coldCandidates, sizeof(Candidate) * height_band * width_band, hipMemcpyDeviceToHost);
  hipMemcpy(&hot_index, d_hot_index, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&cold_index, d_cold_index, sizeof(int), hipMemcpyDeviceToHost);

  std::sort(hotCandidates, hotCandidates + hot_index, compare_candidate_temperature);
  std::sort(coldCandidates, coldCandidates + cold_index, compare_candidate_temperature);

  std::cout << "Hot candidates: " << hot_index << std::endl;
  std::cout << "Cold candidates: " << cold_index << std::endl;

  unsigned int hotPos = static_cast<unsigned int>(std::floor(hot_index * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(cold_index * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

pair<Candidate, Candidate> getEndmembersASEBAL(float *ndvi, float *surface_temperature, float *albedo, float *net_radiation, float *soil_heat, int height_band, int width_band)
{
  vector<Candidate> hotCandidates;
  vector<Candidate> coldCandidates;

  vector<float> tsQuartile(3);
  vector<float> ndviQuartile(3);
  vector<float> albedoQuartile(3);
  get_quartiles(ndvi, ndviQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);
  get_quartiles(albedo, albedoQuartile.data(), height_band, width_band, 0.25, 0.50, 0.75);
  get_quartiles(surface_temperature, tsQuartile.data(), height_band, width_band, 0.25, 0.75, 0.75);

  float *ho = (float *)malloc(sizeof(float) * height_band * width_band);
  compute_H0(net_radiation, soil_heat, height_band, width_band, ho);

  for (int i = 0; i < height_band * width_band; i++)
  {
    bool hotAlbedo = !std::isnan(albedo[i]) && albedo[i] > albedoQuartile[1];
    bool hotNDVI = !std::isnan(ndvi[i]) && ndvi[i] > 0.10 && ndvi[i] < ndviQuartile[0];
    bool hotTS = !std::isnan(surface_temperature[i]) && surface_temperature[i] > tsQuartile[1];

    bool coldAlbedo = !std::isnan(albedo[i]) && albedo[i] < albedoQuartile[1];
    bool coldNDVI = !std::isnan(ndvi[i]) && ndvi[i] >= ndviQuartile[1];
    bool coldTS = !std::isnan(surface_temperature[i]) && surface_temperature[i] < tsQuartile[0];

    int line = i / width_band;
    int col = i % width_band;

    if (hotAlbedo && hotNDVI && hotTS)
      hotCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);

    if (coldNDVI && coldAlbedo && coldTS)
      coldCandidates.emplace_back(ndvi[i], surface_temperature[i], net_radiation[i], soil_heat[i], ho[i], line, col);
  }

  if (hotCandidates.empty() || coldCandidates.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(hotCandidates.begin(), hotCandidates.end(), compare_candidate_temperature);
  std::sort(coldCandidates.begin(), coldCandidates.end(), compare_candidate_temperature);

  unsigned int hotPos = static_cast<unsigned int>(std::floor(hotCandidates.size() * 0.5));
  unsigned int coldPos = static_cast<unsigned int>(std::floor(coldCandidates.size() * 0.5));

  return {hotCandidates[hotPos], coldCandidates[coldPos]};
}

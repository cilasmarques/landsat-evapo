#include "hip/hip_runtime.h"
#include "endmembers.h"

void compute_H0(vector<double> net_radiation_line, vector<double> soil_heat_flux, int width_band, vector<double> &ho_line)
{
  for (int col = 0; col < width_band; col++)
    ho_line[col] = net_radiation_line[col] - soil_heat_flux[col];
};

void filter_valid_values(vector<double> target_line, double *target_values, int width_band, int *pos)
{
  for (int col = 0; col < width_band; col++)
  {
    if (!isnan(target_line[col]) && !isinf(target_line[col]))
    {
      target_values[*pos] = target_line[col];
      (*pos)++;
    }
  }
}

void get_quartiles(vector<vector<double>> target_vector, double *v_quartile, int height_band, int width_band, double first_interval, double last_interval)
{
  const int SIZE = height_band * width_band;
  double *target_values = (double *)malloc(sizeof(double) * SIZE);

  if (target_values == NULL)
    exit(15);

  int pos = 0;
  for (int line = 0; line < height_band; line++)
  {
    filter_valid_values(target_vector[line], target_values, width_band, &pos);
  }

  sort(target_values, target_values + pos);

  v_quartile[0] = target_values[int(floor(first_interval * pos))];
  v_quartile[1] = target_values[int(floor(last_interval * pos))];

  free(target_values);
}

Candidate getHotPixelASEBAL(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> candidatesGroup;
  vector<double> ndviQuartile(2);
  vector<double> tsQuartile(2);
  vector<double> albedoQuartile(2);
  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.25, 0.75);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.25, 0.75);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.25, 0.75);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {

      bool albedoValid = !isnan(albedo_line[col]) && albedo_line[col] > albedoQuartile[1];
      bool ndviValid = !isnan(ndvi_line[col]) && ndvi_line[col] > 0.10 && ndvi_line[col] < ndviQuartile[0];
      bool tsValid = !isnan(surface_temperature_line[col]) && surface_temperature_line[col] > tsQuartile[1];

      if (albedoValid && ndviValid && tsValid)
        candidatesGroup.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (candidatesGroup.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  // Creating second pixel group, all values lower than the 3rd quartile are excluded
  std::sort(candidatesGroup.begin(), candidatesGroup.end(), compare_candidate_temperature);
  unsigned int pos = static_cast<unsigned int>(std::floor(candidatesGroup.size() * 0.75));
  vector<Candidate> candidatesFinalGroup(candidatesGroup.begin() + pos, candidatesGroup.end());

  if (candidatesFinalGroup.size() <= 0)
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  pos = int(floor(candidatesFinalGroup.size() * 0.5));
  Candidate hotPixel = candidatesFinalGroup[pos];

  return hotPixel;
}

Candidate getColdPixelASEBAL(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> candidatesGroup;
  vector<double> ndviQuartile(2);
  vector<double> tsQuartile(2);
  vector<double> albedoQuartile(2);
  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.25, 0.75);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.25, 0.50);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.25, 0.75);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {

      bool albedoValid = !isnan(albedo_line[col]) && albedo_line[col] < albedoQuartile[1];
      bool ndviValid = !isnan(ndvi_line[col]) && ndvi_line[col] >= ndviQuartile[1]; // ndvi_line[col] >= ndviQuartile[3];
      bool tsValid = !isnan(surface_temperature_line[col]) && surface_temperature_line[col] < tsQuartile[0];

      if (albedoValid && ndviValid && tsValid)
        candidatesGroup.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (candidatesGroup.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  // Creating second pixel group, all values lower than the 3rd quartile are excluded
  std::sort(candidatesGroup.begin(), candidatesGroup.end(), compare_candidate_temperature);
  unsigned int pos = static_cast<unsigned int>(std::floor(candidatesGroup.size() * 0.25));
  vector<Candidate> candidatesFinalGroup(candidatesGroup.begin(), candidatesGroup.end() + pos);

  if (candidatesFinalGroup.size() <= 0)
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  pos = int(floor(candidatesFinalGroup.size() * 0.5));
  Candidate coldPixel = candidatesFinalGroup[pos];

  return coldPixel;
}

Candidate getHotPixelSTEPP(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> candidatesGroup;
  vector<double> ndviQuartile(2);
  vector<double> tsQuartile(2);
  vector<double> albedoQuartile(2);
  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.15, 0.85);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.50, 0.75);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.85, 0.97);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {
      bool ndviValid = !std::isnan(ndvi_line[col]) && ndvi_line[col] > 0.10 && ndvi_line[col] < ndviQuartile[0];
      bool albedoValid = !std::isnan(albedo_line[col]) && albedo_line[col] > albedoQuartile[0] && albedo_line[col] < albedoQuartile[1];
      bool tsValid = !std::isnan(surface_temperature_line[col]) && surface_temperature_line[col] > tsQuartile[0] && surface_temperature_line[col] < tsQuartile[1];

      if (albedoValid && ndviValid && tsValid)
        candidatesGroup.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (candidatesGroup.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(candidatesGroup.begin(), candidatesGroup.end(), compare_candidate_temperature);
  unsigned int pos = static_cast<unsigned int>(std::floor(candidatesGroup.size() * 0.5));

  return candidatesGroup[pos];
}

Candidate getColdPixelSTEPP(vector<vector<double>> ndvi_vector, vector<vector<double>> surface_temperature_vector, vector<vector<double>> albedo_vector, vector<vector<double>> net_radiation_vector, vector<vector<double>> soil_heat_vector, int height_band, int width_band)
{
  vector<Candidate> candidatesGroup;
  vector<double> ndviQuartile(2);
  vector<double> tsQuartile(2);
  vector<double> albedoQuartile(2);
  vector<vector<double>> ho_vector(height_band, vector<double>(width_band));

  get_quartiles(ndvi_vector, ndviQuartile.data(), height_band, width_band, 0.15, 0.97);
  get_quartiles(albedo_vector, albedoQuartile.data(), height_band, width_band, 0.25, 0.50);
  get_quartiles(surface_temperature_vector, tsQuartile.data(), height_band, width_band, 0.20, 0.85);

  for (int line = 0; line < height_band; line++)
  {
    vector<double> ho_line = ho_vector[line];
    vector<double> ndvi_line = ndvi_vector[line];
    vector<double> surface_temperature_line = surface_temperature_vector[line];
    vector<double> albedo_line = albedo_vector[line];
    vector<double> net_radiation_line = net_radiation_vector[line];
    vector<double> soil_heat_line = soil_heat_vector[line];

    compute_H0(net_radiation_line, soil_heat_line, width_band, ho_line);

    for (int col = 0; col < width_band; col++)
    {

      bool ndviValid = !std::isnan(ndvi_line[col]) && ndvi_line[col] > ndviQuartile[1];
      bool albedoValid = !std::isnan(surface_temperature_line[col]) && albedo_line[col] > albedoQuartile[0] && albedo_line[col] < albedoQuartile[1];
      bool tsValid = !std::isnan(albedo_line[col]) && surface_temperature_line[col] < tsQuartile[0];

      if (albedoValid && ndviValid && tsValid)
        candidatesGroup.emplace_back(ndvi_line[col], surface_temperature_line[col], net_radiation_line[col], soil_heat_line[col], ho_line[col], line, col);
    }
  }

  if (candidatesGroup.empty())
  {
    cerr << "Pixel problem! - There are no final candidates";
    exit(15);
  }

  std::sort(candidatesGroup.begin(), candidatesGroup.end(), compare_candidate_temperature);
  unsigned int pos = static_cast<unsigned int>(std::floor(candidatesGroup.size() * 0.5));

  return candidatesGroup[pos];
}

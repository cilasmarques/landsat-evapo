#include "hip/hip_runtime.h"
#include "rah_cycle.cuh"

__global__ void correctionCycle(double *surfaceTemperatureLine, double *zomLine, double *ustarRLine, double *ustarWLine, double *rahRLine,
                                double *rahWLine, double *a, double *b, double *u200, int *size)
{
  // Identify position
  int pos = threadIdx.x + blockIdx.x * blockDim.x;

  while (pos < *size)
  {
    double sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (*a + *b * (surfaceTemperatureLine[pos] - 273.15)) / rahRLine[pos];

    double L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustarRLine[pos], 3) * surfaceTemperatureLine[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

    double y01 = pow((1 - (16 * 0.1) / L), 0.25);
    double y2 = pow((1 - (16 * 2) / L), 0.25);
    double x200 = pow((1 - (16 * 200) / L), 0.25);

    double psi01, psi2, psi200;
    if (!isnan(L) && L > 0)
    {
      psi01 = -5 * (0.1 / L);
      psi2 = -5 * (2 / L);
      psi200 = -5 * (2 / L);
    }
    else
    {
      psi01 = 2 * log((1 + y01 * y01) / 2);
      psi2 = 2 * log((1 + y2 * y2) / 2);
      psi200 = 2 * log((1 + x200) / 2) + log((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
    }

    ustarWLine[pos] = (VON_KARMAN * ustarRLine[pos]) / (log(200 / zomLine[pos]) - psi200);
    rahWLine[pos] = (log(2 / 0.1) - psi2 + psi01) / (ustarWLine[pos] * VON_KARMAN);

    pos += blockDim.x * gridDim.x;
  }
}

__global__ void correctionCycleSTEEP(double *surface_temperature_pointer, double *d0_pointer, double *kb1_pointer, double *zom_pointer, double *ustarR_pointer,
                                     double *ustarW_pointer, double *rahR_pointer, double *rahW_pointer, double *H_pointer, double a, double b, int height,
                                     int width)
{
  // Identify position
  unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;

  while (row < height)
  {
    if (col < width && row < height)
    {
      unsigned int pos = row * width + col;

      double DISP = d0_pointer[pos];
      double dT_ini_terra = a + b * (surface_temperature_pointer[pos] - 273.15);

      double sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dT_ini_terra) / rahR_pointer[pos];
      double L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustarR_pointer[pos], 3) * surface_temperature_pointer[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

      double y2 = pow((1 - (16 * (10 - DISP)) / L), 0.25);
      double x200 = pow((1 - (16 * (10 - DISP)) / L), 0.25);

      double psi2, psi200;
      if (!isnan(L) && L > 0)
      {
        psi2 = -5 * ((10 - DISP) / L);
        psi200 = -5 * ((10 - DISP) / L);
      }
      else
      {
        psi2 = 2 * log((1 + y2 * y2) / 2);
        psi200 = 2 * log((1 + x200) / 2) + log((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
      }

      double ust = (VON_KARMAN * ustarR_pointer[pos]) / (log((10 - DISP) / zom_pointer[pos]) - psi200);

      double zoh_terra = zom_pointer[pos] / pow(exp(1.0), (kb1_pointer[pos]));
      double temp_rah1_corr_terra = (ust * VON_KARMAN);
      double temp_rah2_corr_terra = log((10 - DISP) / zom_pointer[pos]) - psi2;
      double temp_rah3_corr_terra = temp_rah1_corr_terra * log(zom_pointer[pos] / zoh_terra);
      double rah = (temp_rah1_corr_terra * temp_rah2_corr_terra) + temp_rah3_corr_terra;

      ustarW_pointer[pos] = ust;
      rahW_pointer[pos] = rah;
      H_pointer[pos] = sensibleHeatFlux;
    }
    row += blockDim.y * gridDim.y;
  }
}
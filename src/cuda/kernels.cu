#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void rad_kernel(float *band_blue_d, float *band_green_d, float *band_red_d, float *band_nir_d, float *band_swir1_d, float *band_termal_d, float *band_swir2_d,
                           float *radiance_blue_d, float *radiance_green_d, float *radiance_red_d, float *radiance_nir_d, float *radiance_swir1_d, float *radiance_termal_d, float *radiance_swir2_d,
                           float *rad_add_d, float *rad_mult_d, int width, int height)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    radiance_blue_d[pos] = band_blue_d[pos] * rad_mult_d[PARAM_BAND_BLUE_INDEX] + rad_add_d[PARAM_BAND_BLUE_INDEX];
    radiance_green_d[pos] = band_green_d[pos] * rad_mult_d[PARAM_BAND_GREEN_INDEX] + rad_add_d[PARAM_BAND_GREEN_INDEX];
    radiance_red_d[pos] = band_red_d[pos] * rad_mult_d[PARAM_BAND_RED_INDEX] + rad_add_d[PARAM_BAND_RED_INDEX];
    radiance_nir_d[pos] = band_nir_d[pos] * rad_mult_d[PARAM_BAND_NIR_INDEX] + rad_add_d[PARAM_BAND_NIR_INDEX];
    radiance_swir1_d[pos] = band_swir1_d[pos] * rad_mult_d[PARAM_BAND_SWIR1_INDEX] + rad_add_d[PARAM_BAND_SWIR1_INDEX];
    radiance_termal_d[pos] = band_termal_d[pos] * rad_mult_d[PARAM_BAND_TERMAL_INDEX] + rad_add_d[PARAM_BAND_TERMAL_INDEX];
    radiance_swir2_d[pos] = band_swir2_d[pos] * rad_mult_d[PARAM_BAND_SWIR2_INDEX] + rad_add_d[PARAM_BAND_SWIR2_INDEX];

    if (radiance_blue_d[pos] <= 0)
      radiance_blue_d[pos] = NAN;
    if (radiance_green_d[pos] <= 0)
      radiance_green_d[pos] = NAN;
    if (radiance_red_d[pos] <= 0)
      radiance_red_d[pos] = NAN;
    if (radiance_nir_d[pos] <= 0)
      radiance_nir_d[pos] = NAN;
    if (radiance_swir1_d[pos] <= 0)
      radiance_swir1_d[pos] = NAN;
    if (radiance_termal_d[pos] <= 0)
      radiance_termal_d[pos] = NAN;
    if (radiance_swir2_d[pos] <= 0)
      radiance_swir2_d[pos] = NAN;
  }
}

__global__ void ref_kernel(float *band_blue_d, float *band_green_d, float *band_red_d, float *band_nir_d, float *band_swir1_d, float *band_termal_d, float *band_swir2_d,
                           float *reflectance_blue_d, float *reflectance_green_d, float *reflectance_red_d, float *reflectance_nir_d, float *reflectance_swir1_d, float *reflectance_termal_d, float *reflectance_swir2_d,
                           float *ref_add_d, float *ref_mult_d, float sin_sun, int width, int height)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    reflectance_blue_d[pos] = (band_blue_d[pos] * ref_mult_d[PARAM_BAND_BLUE_INDEX] + ref_add_d[PARAM_BAND_BLUE_INDEX]) / sin_sun;
    reflectance_green_d[pos] = (band_green_d[pos] * ref_mult_d[PARAM_BAND_GREEN_INDEX] + ref_add_d[PARAM_BAND_GREEN_INDEX]) / sin_sun;
    reflectance_red_d[pos] = (band_red_d[pos] * ref_mult_d[PARAM_BAND_RED_INDEX] + ref_add_d[PARAM_BAND_RED_INDEX]) / sin_sun;
    reflectance_nir_d[pos] = (band_nir_d[pos] * ref_mult_d[PARAM_BAND_NIR_INDEX] + ref_add_d[PARAM_BAND_NIR_INDEX]) / sin_sun;
    reflectance_swir1_d[pos] = (band_swir1_d[pos] * ref_mult_d[PARAM_BAND_SWIR1_INDEX] + ref_add_d[PARAM_BAND_SWIR1_INDEX]) / sin_sun;
    reflectance_termal_d[pos] = (band_termal_d[pos] * ref_mult_d[PARAM_BAND_TERMAL_INDEX] + ref_add_d[PARAM_BAND_TERMAL_INDEX]) / sin_sun;
    reflectance_swir2_d[pos] = (band_swir2_d[pos] * ref_mult_d[PARAM_BAND_SWIR2_INDEX] + ref_add_d[PARAM_BAND_SWIR2_INDEX]) / sin_sun;

    if (reflectance_blue_d[pos] <= 0)
      reflectance_blue_d[pos] = NAN;
    if (reflectance_green_d[pos] <= 0)
      reflectance_green_d[pos] = NAN;
    if (reflectance_red_d[pos] <= 0)
      reflectance_red_d[pos] = NAN;
    if (reflectance_nir_d[pos] <= 0)
      reflectance_nir_d[pos] = NAN;
    if (reflectance_swir1_d[pos] <= 0)
      reflectance_swir1_d[pos] = NAN;
    if (reflectance_termal_d[pos] <= 0)
      reflectance_termal_d[pos] = NAN;
    if (reflectance_swir2_d[pos] <= 0)
      reflectance_swir2_d[pos] = NAN;
  }
}

__global__ void albedo_kernel(float *reflectance_blue_d, float *reflectance_green_d, float *reflectance_red_d, float *reflectance_nir_d, float *reflectance_swir1_d, float *reflectance_swir2_d,
                              float *tal_d, float *albedo_d, float *ref_w_coeff_d, int width, int height)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    float alb_toa = reflectance_blue_d[pos] * ref_w_coeff_d[PARAM_BAND_BLUE_INDEX] +
                    reflectance_green_d[pos] * ref_w_coeff_d[PARAM_BAND_GREEN_INDEX] +
                    reflectance_red_d[pos] * ref_w_coeff_d[PARAM_BAND_RED_INDEX] +
                    reflectance_nir_d[pos] * ref_w_coeff_d[PARAM_BAND_NIR_INDEX] +
                    reflectance_swir1_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR1_INDEX] +
                    reflectance_swir2_d[pos] * ref_w_coeff_d[PARAM_BAND_SWIR2_INDEX];

    albedo_d[pos] = (alb_toa - 0.03) / (tal_d[pos] * tal_d[pos]);

    if (albedo_d[pos] <= 0)
      albedo_d[pos] = NAN;
  }
}

__global__ void ndvi_kernel(float *band_nir_d, float *band_red_d, float *ndvi_d, int width, int height)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    ndvi_d[pos] = (band_nir_d[pos] - band_red_d[pos]) / (band_nir_d[pos] + band_red_d[pos]);

    if (ndvi_d[pos] <= -1 || ndvi_d[pos] >= 1)
      ndvi_d[pos] = NAN;
  }
}

__global__ void pai_kernel(float *band_nir_d, float *band_red_d, float *pai_d, int width, int height)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    pai_d[pos] = 10.1 * (band_nir_d[pos] - sqrt(band_red_d[pos])) + 3.1;

    if (pai_d[pos] <= 0)
      pai_d[pos] = NAN;
  }
}

__global__ void lai_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *lai_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;

    float savi = ((1 + 0.5) * (reflectance_nir_d[pos] - reflectance_red_d[pos])) / (0.5 + (reflectance_nir_d[pos] + reflectance_red_d[pos]));

    if (!isnan(savi) && savi > 0.687)
      lai_d[pos] = 6;
    if (!isnan(savi) && savi <= 0.687)
      lai_d[pos] = -log((0.69 - savi) / 0.59) / 0.91;
    if (!isnan(savi) && savi < 0.1)
      lai_d[pos] = 0;
  }
}

__global__ void evi_kernel(float *reflectance_nir_d, float *reflectance_red_d, float *reflectance_blue_d, float *evi_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;

    evi_d[pos] = 2.5 * ((reflectance_nir_d[pos] - reflectance_red_d[pos]) / (reflectance_nir_d[pos] + (6 * reflectance_red_d[pos]) - (7.5 * reflectance_blue_d[pos]) + 1));

    if (evi_d[pos] < 0)
      evi_d[pos] = NAN;
  }
}

__global__ void enb_kernel(float *lai_d, float *enb_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;

    if (lai_d[pos] == 0)
      enb_d[pos] = NAN;
    else
      enb_d[pos] = 0.97 + 0.0033 * lai_d[pos];

    if (enb_d[pos] < 0 || lai_d[pos] > 2.99)
      enb_d[pos] = 0.98;
  }
}

__global__ void eo_kernel(float *lai_d, float *ndvi_d, float *eo_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;

    if (lai_d[pos] == 0)
      eo_d[pos] = NAN;
    else
      eo_d[pos] = 0.95 + 0.01 * lai_d[pos];

    if (ndvi_d[pos] < 0 || lai_d[pos] > 2.99)
      eo_d[pos] = 0.98;
  }
}

__global__ void ea_kernel(float *tal_d, float *ea_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    ea_d[pos] = 0.85 * pow((-1 * log(tal_d[pos])), 0.09);
  }
}

__global__ void surface_temperature_kernel(float *enb_d, float *radiance_termal_d, float *surface_temperature_d, float k1, float k2, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    surface_temperature_d[pos] = k2 / (log((enb_d[pos] * k1 / radiance_termal_d[pos]) + 1));

    if (surface_temperature_d[pos] < 0)
      surface_temperature_d[pos] = 0;
  }
}

__global__ void short_wave_radiation_kernel(float *tal_d, float *short_wave_radiation_d, float sun_elevation, float distance_earth_sun, float pi, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    short_wave_radiation_d[pos] = (1367 * sin(sun_elevation * pi / 180) * tal_d[pos]) / (distance_earth_sun * distance_earth_sun);
  }
}

__global__ void large_wave_radiation_surface_kernel(float *surface_temperature_d, float *eo_d, float *large_wave_radiation_surface_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    float temperature_pixel = surface_temperature_d[pos];
    float surface_temperature_pow_4 = temperature_pixel * temperature_pixel * temperature_pixel * temperature_pixel;
    large_wave_radiation_surface_d[pos] = eo_d[pos] * 5.67 * 1e-8 * surface_temperature_pow_4;
  }
}

__global__ void large_wave_radiation_atmosphere_kernel(float *ea_d, float *large_wave_radiation_atmosphere_d, float temperature, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    float temperature_kelvin = temperature + 273.15;
    float temperature_kelvin_pow_4 = temperature_kelvin * temperature_kelvin * temperature_kelvin * temperature_kelvin;

    large_wave_radiation_atmosphere_d[pos] = ea_d[pos] * 5.67 * 1e-8 * temperature_kelvin_pow_4;
  }
}

__global__ void net_radiation_kernel(float *short_wave_radiation_d, float *albedo_d, float *large_wave_radiation_atmosphere_d, float *large_wave_radiation_surface_d, float *eo_d, float *net_radiation_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;
    net_radiation_d[pos] = short_wave_radiation_d[pos] - (short_wave_radiation_d[pos] * albedo_d[pos]) + large_wave_radiation_atmosphere_d[pos] - large_wave_radiation_surface_d[pos] - (1 - eo_d[pos]) * large_wave_radiation_atmosphere_d[pos];

    if (net_radiation_d[pos] < 0)
      net_radiation_d[pos] = 0;
  }
}

__global__ void soil_heat_kernel(float *ndvi_d, float *albedo_d, float *surface_temperature_d, float *net_radiation_d, float *soil_heat_d, int width_band, int height_band)
{
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width_band;
  unsigned int col = idx % width_band;

  if (col < width_band && row < height_band)
  {
    unsigned int pos = row * width_band + col;

    if (ndvi_d[pos] < 0 || ndvi_d[pos] > 0)
    {
      float ndvi_pixel_pow_4 = ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos] * ndvi_d[pos];
      soil_heat_d[pos] = (surface_temperature_d[pos] - 273.15) * (0.0038 + 0.0074 * albedo_d[pos]) * (1 - 0.98 * ndvi_pixel_pow_4) * net_radiation_d[pos];
    }
    else
    {
      soil_heat_d[pos] = 0.5 * net_radiation_d[pos];
    }

    if (soil_heat_d[pos] < 0)
      soil_heat_d[pos] = 0;
  }
}

__global__ void rah_correction_cycle_STEEP(float *surface_temperature_pointer, float *d0_pointer, float *kb1_pointer, float *zom_pointer, float *ustarR_pointer,
                                           float *ustarW_pointer, float *rahR_pointer, float *rahW_pointer, float *H_pointer, double a, double b, int height,
                                           int width)
{
  // Identify 1D position
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

  // Map 1D position to 2D grid
  unsigned int row = idx / width;
  unsigned int col = idx % width;

  if (col < width && row < height)
  {
    unsigned int pos = row * width + col;

    double DISP = d0_pointer[pos];
    double dT_ini_terra = a + b * (surface_temperature_pointer[pos] - 273.15);

    double sensibleHeatFlux = RHO * SPECIFIC_HEAT_AIR * (dT_ini_terra) / rahR_pointer[pos];
    double L = -1 * ((RHO * SPECIFIC_HEAT_AIR * pow(ustarR_pointer[pos], 3) * surface_temperature_pointer[pos]) / (VON_KARMAN * GRAVITY * sensibleHeatFlux));

    double y2 = pow((1 - (16 * (10 - DISP)) / L), 0.25);
    double x200 = pow((1 - (16 * (10 - DISP)) / L), 0.25);

    double psi2, psi200;
    if (!isnan(L) && L > 0)
    {
      psi2 = -5 * ((10 - DISP) / L);
      psi200 = -5 * ((10 - DISP) / L);
    }
    else
    {
      psi2 = 2 * log((1 + y2 * y2) / 2);
      psi200 = 2 * log((1 + x200) / 2) + log((1 + x200 * x200) / 2) - 2 * atan(x200) + 0.5 * M_PI;
    }

    double ust = (VON_KARMAN * ustarR_pointer[pos]) / (log((10 - DISP) / zom_pointer[pos]) - psi200);

    double zoh_terra = zom_pointer[pos] / pow(exp(1.0), (kb1_pointer[pos]));
    double temp_rah1_corr_terra = (ust * VON_KARMAN);
    double temp_rah2_corr_terra = log((10 - DISP) / zom_pointer[pos]) - psi2;
    double temp_rah3_corr_terra = temp_rah1_corr_terra * log(zom_pointer[pos] / zoh_terra);
    double rah = (temp_rah1_corr_terra * temp_rah2_corr_terra) + temp_rah3_corr_terra;

    ustarW_pointer[pos] = ust;
    rahW_pointer[pos] = rah;
    H_pointer[pos] = sensibleHeatFlux;
  }
}

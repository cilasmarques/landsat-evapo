#include "hip/hip_runtime.h"
#include "filter.cuh"

__global__ void process_pixels(Candidate *hotCandidates, Candidate *coldCandidates, int *d_indexes,
                               float *ndvi, float *surface_temperature, float *albedo, float *net_radiation, float *soil_heat, float *ho,
                               float ndviQuartileLow, float ndviQuartileHigh, float tsQuartileLow, float tsQuartileMid, float tsQuartileHigh,
                               float albedoQuartileLow, float albedoQuartileMid, float albedoQuartileHigh, int height_band, int width_band)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Map 1D position to 2D grid
    unsigned int row = idx / width_band;
    unsigned int col = idx % width_band;

    if (idx < width_band * height_band)
    {
        unsigned int pos = row * width_band + col;

        ho[pos] = net_radiation[pos] - soil_heat[pos];

        bool hotNDVI = !isnan(ndvi[pos]) && ndvi[pos] > 0.10 && ndvi[pos] < ndviQuartileLow;
        bool hotAlbedo = !isnan(albedo[pos]) && albedo[pos] > albedoQuartileMid && albedo[pos] < albedoQuartileHigh;
        bool hotTS = !isnan(surface_temperature[pos]) && surface_temperature[pos] > tsQuartileMid && surface_temperature[pos] < tsQuartileHigh;

        bool coldNDVI = !isnan(ndvi[pos]) && ndvi[pos] > ndviQuartileHigh;
        bool coldAlbedo = !isnan(surface_temperature[pos]) && albedo[pos] > albedoQuartileLow && albedo[pos] < albedoQuartileMid;
        bool coldTS = !isnan(albedo[pos]) && surface_temperature[pos] < tsQuartileLow;

        if (hotAlbedo && hotNDVI && hotTS)
        {
            int ih = atomicAdd(&d_indexes[0],1);
            hotCandidates[ih] = Candidate(ndvi[pos], surface_temperature[pos], net_radiation[pos], soil_heat[pos], ho[pos], row, col);
        }

        if (coldNDVI && coldAlbedo && coldTS)
        {
            int ic = atomicAdd(&d_indexes[1],1);
            coldCandidates[ic] = Candidate(ndvi[pos], surface_temperature[pos], net_radiation[pos], soil_heat[pos], ho[pos], row, col);
        }
    }
}

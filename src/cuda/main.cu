#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "landsat.h"
#include "constants.h"
#include "parameters.h"
#include "reader.h"

using namespace std;

/** 
 * @brief Main function
 * This function is responsible for reading the input parameters and calling the Landsat class to process the products.
 * 
 * @param argc Number of input parameters
 * @param argv Input parameters
 *              - INPUT_BAND_BLUE_INDEX    = 1;
 *              - INPUT_BAND_GREEN_INDEX   = 2;
 *              - INPUT_BAND_RED_INDEX     = 3;
 *              - INPUT_BAND_NIR_INDEX     = 4;
 *              - INPUT_BAND_SWIR1_INDEX   = 5;
 *              - INPUT_BAND_TERMAL_INDEX  = 6;
 *              - INPUT_BAND_SWIR2_INDEX   = 7;
 *              - INPUT_BAND_TAL_INDEX     = 8;
 *              - INPUT_MTL_DATA_INDEX     = 9;
 *              - INPUT_STATION_DATA_INDEX = 10;
 *              - INPUT_LAND_COVER_INDEX   = 11;
 *              - OUTPUT_FOLDER            = 12;
 * @return int
*/
int main(int argc, char *argv[])
{
  int INPUT_BAND_TAL_INDEX    = 8;
  int INPUT_MTL_DATA_INDEX     = 9;
  int INPUT_STATION_DATA_INDEX = 10;
  int INPUT_LAND_COVER_INDEX   = 11;
  int OUTPUT_FOLDER            = 12;

  // load meta data
  string path_meta_file = argv[INPUT_MTL_DATA_INDEX];
  MTL mtl = MTL(path_meta_file);

  // load station data
  string station_data_path = argv[INPUT_STATION_DATA_INDEX];
  Station station = Station(station_data_path, mtl.image_hour);

  // load sensor data
  Sensor sensor = Sensor(mtl.number_sensor, mtl.year);

  // load bands path
  string bands_paths[INPUT_BAND_TAL_INDEX];
  for (int i = 1; i < INPUT_BAND_TAL_INDEX; i++) {
    bands_paths[i] = argv[i];
  }

  // load tal path
  string tal_path = argv[INPUT_BAND_TAL_INDEX];

  // load land cover path
  string land_cover_path = argv[INPUT_LAND_COVER_INDEX];

  // load selected method 
  int method = 0;
  if(argc >= 14){
    string flag = argv[13];
    if(flag.substr(0, 6) == "-meth=")
      method = flag[6] - '0';
  }

  // load threads number
  int threads_num = 8;
  if(argc >= 15){
    string threads_flag = argv[14];
    if(threads_flag.substr(0,9) == "-threads=")
      threads_num = atof(threads_flag.substr(9, threads_flag.size()).c_str());
  }

  // load blocks number
  int blocks_num = 6504;
  if(argc >= 16){
    string blocks_flag = argv[15];
    if(blocks_flag.substr(0,8) == "-blocks=")
      blocks_num = atof(blocks_flag.substr(8, blocks_flag.size()).c_str());
  }

  //Timing
  using namespace std::chrono;
  system_clock::time_point begin, end;
  int64_t initial_time, final_time, general_time;
  std::cout << "PHASE,TIMESTAMP,START_TIME,END_TIME" << std::endl;
  initial_time = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
  begin = system_clock::now();

  Landsat landsat = Landsat(method, bands_paths, tal_path, land_cover_path, threads_num, blocks_num);
  landsat.process_products(mtl, sensor, station);

  end = system_clock::now();
  final_time = duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
  general_time = duration_cast<milliseconds>(end.time_since_epoch() - begin.time_since_epoch()).count();
  std::cout << "TOTAL," << general_time << "," << initial_time << "," << final_time << std::endl;

  // =====  END + OUTPUTS =====

  string outPath = std::string(argv[OUTPUT_FOLDER]) + "/metadata.txt";
  std::ofstream outputThreads(outPath);
  std::streambuf* coutThreads = std::cout.rdbuf();
  std::cout.rdbuf(outputThreads.rdbuf());
  std::cout << outPath << std::endl;
  std::cout << "informed threads: " << threads_num << std::endl;
  std::cout << "informed blocks: " << blocks_num << std::endl;

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int device = 0; device < deviceCount; ++device) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, device);
      
      std::cout << "Device " << device << " is a " << deviceProp.name << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.multiProcessorCount << " SMs" << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.persistingL2CacheMaxSize  << " bytes of L2 cache" << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.concurrentKernels << " concurrent kernels" << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.maxBlocksPerMultiProcessor << " max blocks per SM" << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.maxThreadsPerMultiProcessor << " max threads per SM" << std::endl;
      std::cout << "Device " << device << " has " << deviceProp.maxGridSize << " max grid size" << std::endl;
  }

  return 0;
}

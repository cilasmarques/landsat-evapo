#include "hip/hip_runtime.h"
#include "products.h"
#include "kernels.cuh"

Products::Products() {}

Products::Products(uint32_t width_band, uint32_t height_band, int threads_num)
{
  this->threads_num = threads_num;
  this->blocks_num = ceil(width_band * height_band / this->threads_num);

  this->width_band = width_band;
  this->height_band = height_band;
  this->nBytes_band = height_band * width_band * sizeof(float);

  this->band_blue = (float *)malloc(nBytes_band);
  this->band_green = (float *)malloc(nBytes_band);
  this->band_red = (float *)malloc(nBytes_band);
  this->band_nir = (float *)malloc(nBytes_band);
  this->band_swir1 = (float *)malloc(nBytes_band);
  this->band_termal = (float *)malloc(nBytes_band);
  this->band_swir2 = (float *)malloc(nBytes_band);
  this->tal = (float *)malloc(nBytes_band);

  this->radiance_blue = (float *)malloc(nBytes_band);
  this->radiance_green = (float *)malloc(nBytes_band);
  this->radiance_red = (float *)malloc(nBytes_band);
  this->radiance_nir = (float *)malloc(nBytes_band);
  this->radiance_swir1 = (float *)malloc(nBytes_band);
  this->radiance_termal = (float *)malloc(nBytes_band);
  this->radiance_swir2 = (float *)malloc(nBytes_band);

  this->reflectance_blue = (float *)malloc(nBytes_band);
  this->reflectance_green = (float *)malloc(nBytes_band);
  this->reflectance_red = (float *)malloc(nBytes_band);
  this->reflectance_nir = (float *)malloc(nBytes_band);
  this->reflectance_swir1 = (float *)malloc(nBytes_band);
  this->reflectance_termal = (float *)malloc(nBytes_band);
  this->reflectance_swir2 = (float *)malloc(nBytes_band);

  this->albedo = (float *)malloc(nBytes_band);
  this->ndvi = (float *)malloc(nBytes_band);
  this->soil_heat = (float *)malloc(nBytes_band);
  this->surface_temperature = (float *)malloc(nBytes_band);
  this->net_radiation = (float *)malloc(nBytes_band);
  this->lai = (float *)malloc(nBytes_band);
  this->savi = (float *)malloc(nBytes_band);
  this->evi = (float *)malloc(nBytes_band);
  this->pai = (float *)malloc(nBytes_band);
  this->enb_emissivity = (float *)malloc(nBytes_band);
  this->eo_emissivity = (float *)malloc(nBytes_band);
  this->ea_emissivity = (float *)malloc(nBytes_band);
  this->short_wave_radiation = (float *)malloc(nBytes_band);
  this->large_wave_radiation_surface = (float *)malloc(nBytes_band);
  this->large_wave_radiation_atmosphere = (float *)malloc(nBytes_band);

  this->surface_temperature = (float *)malloc(nBytes_band);
  this->d0 = (float *)malloc(nBytes_band);
  this->zom = (float *)malloc(nBytes_band);
  this->ustar = (float *)malloc(nBytes_band);
  this->kb1 = (float *)malloc(nBytes_band);
  this->aerodynamic_resistance = (float *)malloc(nBytes_band);
  this->sensible_heat_flux = (float *)malloc(nBytes_band);

  this->latent_heat_flux = (float *)malloc(nBytes_band);
  this->net_radiation_24h = (float *)malloc(nBytes_band);
  this->evapotranspiration_fraction = (float *)malloc(nBytes_band);
  this->sensible_heat_flux_24h = (float *)malloc(nBytes_band);
  this->latent_heat_flux_24h = (float *)malloc(nBytes_band);
  this->evapotranspiration_24h = (float *)malloc(nBytes_band);
  this->evapotranspiration = (float *)malloc(nBytes_band);

  HANDLE_ERROR(hipMalloc((void **)&this->band_blue_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_green_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_red_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_nir_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_swir1_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_termal_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->band_swir2_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->tal_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->radiance_blue_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_green_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_red_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_nir_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_swir1_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_termal_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->radiance_swir2_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_blue_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_green_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_red_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_nir_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_swir1_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_termal_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->reflectance_swir2_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->albedo_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->ndvi_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->pai_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->lai_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->evi_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->enb_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->eo_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->ea_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->short_wave_radiation_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->large_wave_radiation_surface_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->large_wave_radiation_atmosphere_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->surface_temperature_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->net_radiation_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->soil_heat_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->zom_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->d0_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->kb1_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->ts_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->ustarR_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->ustarW_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->rahR_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->rahW_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->sensible_heat_flux_d, nBytes_band));

  HANDLE_ERROR(hipMalloc((void **)&this->latent_heat_flux_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->net_radiation_24h_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->evapotranspiration_fraction_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->sensible_heat_flux_24h_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->latent_heat_flux_24h_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->evapotranspiration_24h_d, nBytes_band));
  HANDLE_ERROR(hipMalloc((void **)&this->evapotranspiration_d, nBytes_band));
};

void Products::close()
{
  free(band_blue);
  free(band_green);
  free(band_red);
  free(band_nir);
  free(band_swir1);
  free(band_termal);
  free(band_swir2);
  free(tal);

  free(this->radiance_blue);
  free(this->radiance_green);
  free(this->radiance_red);
  free(this->radiance_nir);
  free(this->radiance_swir1);
  free(this->radiance_termal);
  free(this->radiance_swir2);

  free(this->reflectance_blue);
  free(this->reflectance_green);
  free(this->reflectance_red);
  free(this->reflectance_nir);
  free(this->reflectance_swir1);
  free(this->reflectance_termal);
  free(this->reflectance_swir2);

  free(this->albedo);
  free(this->ndvi);
  free(this->soil_heat);
  free(this->surface_temperature);
  free(this->net_radiation);
  free(this->lai);
  free(this->evi);
  free(this->pai);
  free(this->enb_emissivity);
  free(this->eo_emissivity);
  free(this->ea_emissivity);
  free(this->short_wave_radiation);
  free(this->large_wave_radiation_surface);
  free(this->large_wave_radiation_atmosphere);

  free(this->surface_temperature);
  free(this->d0);
  free(this->zom);
  free(this->ustar);
  free(this->kb1);
  free(this->aerodynamic_resistance);
  free(this->sensible_heat_flux);

  free(this->latent_heat_flux);
  free(this->net_radiation_24h);
  free(this->evapotranspiration_fraction);
  free(this->sensible_heat_flux_24h);
  free(this->latent_heat_flux_24h);
  free(this->evapotranspiration_24h);
  free(this->evapotranspiration);
}

void Products::radiance_function(MTL mtl)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  rad_kernel<<<this->blocks_num, this->threads_num>>>(band_blue_d, band_green_d, band_red_d, band_nir_d, band_swir1_d, band_termal_d, band_swir2_d,
                                                      radiance_blue_d, radiance_green_d, radiance_red_d, radiance_nir_d, radiance_swir1_d, radiance_termal_d, radiance_swir2_d,
                                                      mtl.rad_add_d, mtl.rad_mult_d, width_band, height_band);
  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,RADIANCE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  HANDLE_ERROR(hipMemcpy(radiance_blue, radiance_blue_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_green, radiance_green_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_red, radiance_red_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_nir, radiance_nir_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_swir1, radiance_swir1_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_termal, radiance_termal_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(radiance_swir2, radiance_swir2_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
}

void Products::reflectance_function(MTL mtl)
{
  const float sin_sun = sin(mtl.sun_elevation * PI / 180);

  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  ref_kernel<<<this->blocks_num, this->threads_num>>>(band_blue_d, band_green_d, band_red_d, band_nir_d, band_swir1_d, band_termal_d, band_swir2_d,
                                                      reflectance_blue_d, reflectance_green_d, reflectance_red_d, reflectance_nir_d, reflectance_swir1_d, reflectance_termal_d, reflectance_swir2_d,
                                                      mtl.ref_add_d, mtl.ref_mult_d, sin_sun, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,REFLECTANCE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(reflectance_blue, reflectance_blue_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_green, reflectance_green_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_red, reflectance_red_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_nir, reflectance_nir_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_swir1, reflectance_swir1_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_termal, reflectance_termal_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(reflectance_swir2, reflectance_swir2_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
}

void Products::albedo_function(MTL mtl)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  albedo_kernel<<<this->blocks_num, this->threads_num>>>(reflectance_blue_d, reflectance_green_d, reflectance_red_d, reflectance_nir_d, reflectance_swir1_d, reflectance_swir2_d,
                                                         tal_d, albedo_d, mtl.ref_w_coeff_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,ALBEDO," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(albedo, albedo_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
}

void Products::ndvi_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  ndvi_kernel<<<this->blocks_num, this->threads_num>>>(reflectance_nir_d, reflectance_red_d, ndvi_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,NDVI," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(ndvi, ndvi_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::pai_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  pai_kernel<<<this->blocks_num, this->threads_num>>>(reflectance_nir_d, reflectance_red_d, pai_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,PAI," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(pai, pai_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::lai_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  lai_kernel<<<this->blocks_num, this->threads_num>>>(reflectance_nir_d, reflectance_red_d, lai_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,LAI," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(lai, lai_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::evi_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  evi_kernel<<<this->blocks_num, this->threads_num>>>(reflectance_nir_d, reflectance_red_d, reflectance_blue_d, evi_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EVI," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(evi, evi_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::enb_emissivity_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  enb_kernel<<<this->blocks_num, this->threads_num>>>(lai_d, enb_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,ENB," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(enb_emissivity, enb_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::eo_emissivity_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  eo_kernel<<<this->blocks_num, this->threads_num>>>(lai_d, ndvi_d, eo_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EO," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(eo_emissivity, eo_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::ea_emissivity_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  ea_kernel<<<this->blocks_num, this->threads_num>>>(tal_d, ea_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EA," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(ea_emissivity, ea_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::surface_temperature_function(MTL mtl)
{
  double k1, k2;
  switch (mtl.number_sensor)
  {
  case 5:
    k1 = 607.76;
    k2 = 1260.56;
    break;

  case 7:
    k1 = 666.09;
    k2 = 1282.71;
    break;

  case 8:
    k1 = 774.8853;
    k2 = 1321.0789;
    break;

  default:
    cerr << "Sensor problem!";
    exit(6);
  }

  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  surface_temperature_kernel<<<this->blocks_num, this->threads_num>>>(enb_d, radiance_termal_d, surface_temperature_d, k1, k2, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,SURFACE_TEMPERATURE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(surface_temperature, surface_temperature_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::short_wave_radiation_function(MTL mtl)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  short_wave_radiation_kernel<<<this->blocks_num, this->threads_num>>>(tal_d, short_wave_radiation_d, mtl.sun_elevation, mtl.distance_earth_sun, PI, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,SHORT_WAVE_RADIATION," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(short_wave_radiation, short_wave_radiation_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::large_wave_radiation_surface_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  large_wave_radiation_surface_kernel<<<this->blocks_num, this->threads_num>>>(surface_temperature_d, eo_d, large_wave_radiation_surface_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,LARGE_WAVE_RADIATION_SURFACE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(large_wave_radiation_surface, large_wave_radiation_surface_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::large_wave_radiation_atmosphere_function(double temperature)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  large_wave_radiation_atmosphere_kernel<<<this->blocks_num, this->threads_num>>>(ea_d, large_wave_radiation_atmosphere_d, temperature, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,LARGE_WAVE_RADIATION_ATMOSPHERE," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(large_wave_radiation_atmosphere, large_wave_radiation_atmosphere_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::net_radiation_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  net_radiation_kernel<<<this->blocks_num, this->threads_num>>>(short_wave_radiation_d, albedo_d, large_wave_radiation_atmosphere_d, large_wave_radiation_surface_d, eo_d, net_radiation_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,NET_RADIATION," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(net_radiation, net_radiation_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::soil_heat_flux_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  soil_heat_kernel<<<this->blocks_num, this->threads_num>>>(ndvi_d, albedo_d, surface_temperature_d, net_radiation_d, soil_heat_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,SOIL_HEAT_FLUX," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(soil_heat, soil_heat_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::d0_fuction()
{
  float CD1 = 20.6;
  float HGHT = 4;

  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  d0_kernel<<<this->blocks_num, this->threads_num>>>(pai_d, d0_d, CD1, HGHT, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,D0," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(d0, d0_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::zom_fuction(double A_ZOM, double B_ZOM)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  zom_kernel<<<this->blocks_num, this->threads_num>>>(d0_d, pai_d, zom_d, A_ZOM, B_ZOM, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,ZOM," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(zom, zom_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::ustar_fuction(double u10)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  ustar_kernel<<<this->blocks_num, this->threads_num>>>(zom_d, d0_d, ustarR_d, u10, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,USTAR," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(ustar, ustarR_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::kb_function(double ndvi_max, double ndvi_min)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  kb_kernel<<<this->blocks_num, this->threads_num>>>(zom_d, ustarR_d, pai_d, kb1_d, ndvi_d, width_band, height_band, ndvi_max, ndvi_min);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,KB," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(kb1, kb1_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::aerodynamic_resistance_fuction()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  aerodynamic_resistance_kernel<<<this->blocks_num, this->threads_num>>>(zom_d, d0_d, ustarR_d, kb1_d, rahR_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,RAH_INI," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(aerodynamic_resistance, rahR_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::sensible_heat_flux_function(double a, double b)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  sensible_heat_flux_kernel<<<this->blocks_num, this->threads_num>>>(surface_temperature_d, rahR_d, net_radiation_d, soil_heat_d, sensible_heat_flux_d, a, b, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,SENSIBLE_HEAT_FLUX," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(sensible_heat_flux, sensible_heat_flux_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::latent_heat_flux_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  latent_heat_flux_kernel<<<this->blocks_num, this->threads_num>>>(net_radiation_d, soil_heat_d, sensible_heat_flux_d, latent_heat_flux_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,LATENT_HEAT_FLUX," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(latent_heat_flux, latent_heat_flux_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::net_radiation_24h_function(double Ra24h, double Rs24h)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  net_radiation_24h_kernel<<<this->blocks_num, this->threads_num>>>(albedo_d, Rs24h, Ra24h, net_radiation_24h_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,NET_RADIATION_24H," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(net_radiation_24h, net_radiation_24h_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::evapotranspiration_fraction_fuction()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  evapotranspiration_fraction_kernel<<<this->blocks_num, this->threads_num>>>(net_radiation_d, soil_heat_d, latent_heat_flux_d, evapotranspiration_fraction_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EVAPOTRANSPIRATION_FRACTION," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(evapotranspiration_fraction, evapotranspiration_fraction_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::sensible_heat_flux_24h_fuction()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  sensible_heat_flux_24h_kernel<<<this->blocks_num, this->threads_num>>>(net_radiation_24h_d, evapotranspiration_fraction_d, sensible_heat_flux_24h_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,SENSIBLE_HEAT_FLUX_24H," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(sensible_heat_flux_24h, sensible_heat_flux_24h_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::latent_heat_flux_24h_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  latent_heat_flux_24h_kernel<<<this->blocks_num, this->threads_num>>>(net_radiation_24h_d, evapotranspiration_fraction_d, latent_heat_flux_24h_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,LATENT_HEAT_FLUX_24H," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(latent_heat_flux_24h, latent_heat_flux_24h_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::evapotranspiration_24h_function(Station station)
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  evapotranspiration_24h_kernel<<<this->blocks_num, this->threads_num>>>(latent_heat_flux_24h_d, evapotranspiration_24h_d, station.v7_max, station.v7_min, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EVAPOTRANSPIRATION_24H," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(evapotranspiration_24h, evapotranspiration_24h_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

void Products::evapotranspiration_function()
{
  system_clock::time_point begin, end;
  int64_t general_time, initial_time, final_time;

  begin = system_clock::now();
  initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

  evapotranspiration_kernel<<<this->blocks_num, this->threads_num>>>(net_radiation_24h_d, evapotranspiration_fraction_d, evapotranspiration_d, width_band, height_band);

  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipGetLastError());

  end = system_clock::now();
  general_time = duration_cast<nanoseconds>(end - begin).count();
  final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
  std::cout << "CUDACORE,EVAPOTRANSPIRATION," + std::to_string(general_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";

  // Copy data back to host
  HANDLE_ERROR(hipMemcpy(evapotranspiration, evapotranspiration_d, sizeof(float) * height_band * width_band, hipMemcpyDeviceToHost));
};

string Products::rah_correction_function_blocks(double ndvi_min, double ndvi_max, Candidate hot_pixel, Candidate cold_pixel)
{
  system_clock::time_point begin_core, end_core;
  int64_t general_time_core, initial_time_core, final_time_core;

  // ========= CUDA Setup
  int dev = 0;
  hipDeviceProp_t deviceProp;
  HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, dev));
  HANDLE_ERROR(hipSetDevice(dev));

  int threads_per_block = threads_num;
  int num_blocks = ceil(width_band * height_band / threads_per_block);

  double hot_pixel_aerodynamic = aerodynamic_resistance[hot_pixel.line * width_band + hot_pixel.col];
  hot_pixel.aerodynamic_resistance.push_back(hot_pixel_aerodynamic);

  double cold_pixel_aerodynamic = aerodynamic_resistance[cold_pixel.line * width_band + cold_pixel.col];
  cold_pixel.aerodynamic_resistance.push_back(cold_pixel_aerodynamic);

  double fc_hot = 1 - pow((ndvi[hot_pixel.line * width_band + hot_pixel.col] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);
  double fc_cold = 1 - pow((ndvi[cold_pixel.line * width_band + cold_pixel.col] - ndvi_max) / (ndvi_min - ndvi_max), 0.4631);

  for (int i = 0; i < 2; i++)
  {
    this->rah_ini_pq_terra = hot_pixel.aerodynamic_resistance[i];
    this->rah_ini_pf_terra = cold_pixel.aerodynamic_resistance[i];

    double LEc_terra = 0.55 * fc_hot * (hot_pixel.net_radiation - hot_pixel.soil_heat_flux) * 0.78;
    double LEc_terra_pf = 1.75 * fc_cold * (cold_pixel.net_radiation - cold_pixel.soil_heat_flux) * 0.78;

    this->H_pf_terra = cold_pixel.net_radiation - cold_pixel.soil_heat_flux - LEc_terra_pf;
    double dt_pf_terra = H_pf_terra * rah_ini_pf_terra / (RHO * SPECIFIC_HEAT_AIR);

    this->H_pq_terra = hot_pixel.net_radiation - hot_pixel.soil_heat_flux - LEc_terra;
    double dt_pq_terra = H_pq_terra * rah_ini_pq_terra / (RHO * SPECIFIC_HEAT_AIR);

    double b = (dt_pq_terra - dt_pf_terra) / (hot_pixel.temperature - cold_pixel.temperature);
    double a = dt_pf_terra - (b * (cold_pixel.temperature - 273.15));

    HANDLE_ERROR(hipMemcpy(ts_d, surface_temperature, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(zom_d, zom, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d0_d, d0, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(kb1_d, kb1, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(ustarR_d, ustar, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(rahR_d, aerodynamic_resistance, nBytes_band, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(sensible_heat_flux_d, sensible_heat_flux, nBytes_band, hipMemcpyHostToDevice)); // An empty array to receive the results

    // ==== Paralelization core
    begin_core = system_clock::now();
    initial_time_core = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    rah_correction_cycle_STEEP<<<num_blocks, threads_per_block>>>(ts_d, d0_d, kb1_d, zom_d, ustarR_d, ustarW_d, rahR_d, rahW_d, sensible_heat_flux_d, a, b, height_band, width_band);
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipGetLastError());

    end_core = system_clock::now();
    general_time_core = duration_cast<nanoseconds>(end_core - begin_core).count();
    final_time_core = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();
    // ====

    HANDLE_ERROR(hipMemcpy(ustar, ustarW_d, nBytes_band, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(aerodynamic_resistance, rahW_d, nBytes_band, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(sensible_heat_flux, sensible_heat_flux_d, nBytes_band, hipMemcpyDeviceToHost));

    double rah_hot = this->aerodynamic_resistance[hot_pixel.line * width_band + hot_pixel.col];
    hot_pixel.aerodynamic_resistance.push_back(rah_hot);

    double rah_cold = this->aerodynamic_resistance[cold_pixel.line * width_band + cold_pixel.col];
    cold_pixel.aerodynamic_resistance.push_back(rah_cold);
  }

  return "P2 - RAH - PARALLEL - CORE, " + to_string(general_time_core) + ", " + to_string(initial_time_core) + ", " + to_string(final_time_core) + "\n";
}
#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "kernels.cuh"
#include "sensors.cuh"
#include "surfaceData.cuh"

string latent_heat_flux_function(Products products)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    latent_heat_flux_kernel<<<blocks_n, threads_n>>>(products.net_radiation_d, products.soil_heat_d, products.sensible_heat_flux_d, products.latent_heat_flux_d);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,LATENT_HEAT_FLUX," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string net_radiation_24h_function(Products products, Station station, MTL mtl)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    float dr = (1 / mtl.distance_earth_sun) * (1 / mtl.distance_earth_sun);
    float sigma = 0.409 * sin(((2 * PI / 365) * mtl.julian_day) - 1.39);
    float phi = (PI / 180) * station.latitude;
    float omegas = acos(-tan(phi) * tan(sigma));
    float Ra24h = (((24 * 60 / PI) * GSC * dr) * (omegas * sin(phi) * sin(sigma) + cos(phi) * cos(sigma) * sin(omegas))) * (1000000 / 86400.0);
    float Rs24h = station.INTERNALIZATION_FACTOR * sqrt(station.v7_max - station.v7_min) * Ra24h;
    net_radiation_24h_kernel<<<blocks_n, threads_n>>>(products.albedo_d, Rs24h, Ra24h, products.net_radiation_24h_d);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,NET_RADIATION_24H," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string evapotranspiration_24h_function(Products products, Station station)
{
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    evapotranspiration_24h_kernel<<<blocks_n, threads_n>>>(products.surface_temperature_d, products.latent_heat_flux_d, products.net_radiation_d, products.soil_heat_d, products.net_radiation_24h_d, products.evapotranspiration_24h_d);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    return "KERNELS,EVAPOTRANSPIRATION_24H," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
};

string Products::compute_H_ET(Products products, Station station, MTL mtl)
{
    string result = "";
    int64_t initial_time, final_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    initial_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    hipEventRecord(start);
    result += latent_heat_flux_function(products);
    result += net_radiation_24h_function(products, station, mtl);
    result += evapotranspiration_24h_function(products, station);
    hipEventRecord(stop);

    float cuda_time = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cuda_time, start, stop);
    final_time = duration_cast<nanoseconds>(system_clock::now().time_since_epoch()).count();

    result += "KERNELS,P4_FINAL_PROD," + std::to_string(cuda_time) + "," + std::to_string(initial_time) + "," + std::to_string(final_time) + "\n";
    return result;
};
